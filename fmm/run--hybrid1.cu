
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#include "evaluate.h"
#include "util.h"
#include "reals.h"
#include "partial.h"
#include "../timing/timing.h"
#include "node_gpu.h"
#include <cutil_inline.h>

int
run (FMMWrapper_t *F)
{
  struct stopwatch_t* timer = NULL;
  long double t_total;
  timer = stopwatch_create ();

    fprintf (stderr, "Performing Ulist on the GPU\n");
  stopwatch_start (timer);
    ulist_calc__gpu (F);
    up_calc__cpu (F);
    vlist_calc__cpu (F);
		// wlist_calc__cpu (F);
		// xlist_calc__cpu (F);
    down_calc__cpu (F);
    hipDeviceSynchronize ();
    copy_trg_val__gpu (F);
  t_total = stopwatch_stop (timer);
  
  fprintf(stderr, "=== Statistics ===\n");
  fprintf (stdout, "Code: %s\n", get_implementation_name ());
  fprintf (stdout, "Floating-point word size: %lu bytes\n", sizeof (real_t));
  fprintf (stderr, "== Timing for FMM3d ==\n");
  fprintf (stderr, "  ==> Total Execution Time: %Lg secs\n", t_total);

  stopwatch_destroy (timer);

  return 0;
}
/* ------------------------------------------------------------------------
 */
