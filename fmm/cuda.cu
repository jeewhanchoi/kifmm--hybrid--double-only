#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <cutil_inline.h>

#include "evaluate.h"
#include "util.h"
#include "reals.h"
#include "../timing/timing.h"
#include "node_gpu.h"

#define NP_3 0
#define NP_4 0
#define NP_6 1

#include "partial.h"

  void
gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n",
        filename, line,
        C_E, hipGetErrorString (C_E));
    fflush (fp);
    exit (-1); /* abort program */
  }
}

/* ------------------------------------------------------------------------
 */
  int
get_thread_block_size_ulist()
{
  return getenv__int("TBSIZE_ULIST", 128);
}

/* ------------------------------------------------------------------------
 */
  int
get_thread_block_size_up()
{
  return getenv__int("TBSIZE_UP", 128);
}

/* ------------------------------------------------------------------------
 */

  int
get_thread_block_size_up_reduce()
{
	if(NP_3) {
	  return getenv__int("TBSIZE_UP_REDUCE", 256);
	} else if (NP_4) {
	  return getenv__int("TBSIZE_UP_REDUCE", 256);
	} else if (NP_6) {
	  return getenv__int("TBSIZE_UP_REDUCE", 256);
	}
}

/* ------------------------------------------------------------------------
 */

  int
get_thread_block_size_fft_trans()
{
  return getenv__int("TBSIZE_FFT_TRANS", 256);
}

/* ------------------------------------------------------------------------
 */

  int
get_thread_block_size_vlist()
{
  return getenv__int("TBSIZE_VLIST", 128);
}

/* ------------------------------------------------------------------------
 */

  int
get_thread_block_size_down()
{
  return getenv__int("TBSIZE_DOWN", 256);
}

/* ------------------------------------------------------------------------
 */

  int
get_thread_block_size_down_leaf()
{
  return getenv__int("TBSIZE_DOWN_LEAF", 128);
}
/* ------------------------------------------------------------------------
 */

int
get_thread_block_size_wlist()
{
  return getenv__int("TBSIZE_WLIST", 256);
}


__global__
  void
up_eval__gpu (int n_boxes_, int *Bptr_, int *Bn_, 
    dtype *x_, dtype *y_, dtype *z_, dtype *w_,
    dtype *radius_, dtype *c0_, dtype *c1_, dtype *c2_,
    int sp_uc_size, int sp_uc_size_padded, dtype* sp_uc_,
    int uc2ue_r, int uc2ue_r_padded, int uc2ue_c, dtype *uc2ue_,
    /* int num_non_leaf_nodes, */ int *depth_,
    dtype* src_upw_equ_den_)
{
  int i, j;
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

  if(bid < n_boxes_) {
    int start, end;

    /* beginning and ending points for this box/node */
    start = Bptr_[bid];
    end = Bptr_[bid] + Bn_[bid];

    /* do it only if this box/node is a leaf node */
    if(start < end) {
      __shared__ dtype potential[BLK_SIZE_UP];
      /* each thread is responsible for at least 1 point in tl_pos */
      /* there are SP[UC].n points in tl_pos */
      /* for each point in tl_pos, iterate over all source points
         corresponding to this source leaf node */
      for(i = tid; i < sp_uc_size; i += blockDim.x) {
        /* there are variables required for each point in tl_pos */
        dtype c0 = c0_[bid];
        dtype c1 = c1_[bid];
        dtype c2 = c2_[bid];
        dtype r = radius_[bid];

        dtype sp_x = sp_uc_[i];
        dtype sp_y = sp_uc_[1 * sp_uc_size_padded + i];
        dtype sp_z = sp_uc_[2 * sp_uc_size_padded + i];

        sp_x = c0 + r * sp_x;
        sp_y = c1 + r * sp_y;
        sp_z = c2 + r * sp_z;
        dtype sp_d = 0.0;

        /* now loop over all points in source leaf node */
        for(j = start; j < end; j++) {
          /* load source's x, y, z, and w */
          dtype x = x_[j]; 
          dtype y = y_[j]; 
          dtype z = z_[j]; 
          dtype w = w_[j];

          x = sp_x - x;
          y = sp_y - y;
          z = sp_z - z;
          dtype rsq = (x * x) + (y * y) + (z * z);
          rsq = rsqrt (rsq);
          sp_d += w * rsq;
        }
        potential[i] = OOFP_R * sp_d;
      }
      __syncthreads();

		
      /* do dgemv */
      dtype td = 0.0;
      dtype sclvec = depth_[bid];
      sclvec = __powf (2.0, -1 * sclvec); /* it's okay to do this in float */
      for(i = tid; i < uc2ue_r; i += blockDim.x) {
        td = 0.0;
        for(j = 0; j < uc2ue_c; j++) {
          td += uc2ue_[j * uc2ue_r_padded + i] * potential[j];
        }


        /* scale */
        td = td * sclvec;
        src_upw_equ_den_[bid * uc2ue_r_padded + i] = td;
      }

    }
  }
}

/* ------------------------------------------------------------------------
 */

__global__
  void
up_eval__gpu_reduction (int num_thr_per_child, 
    int n_boxes_, int reduction_offset, int node_depth,
    int *children,
    dtype *src_upw_equ_den_, int uc2ue_r_padded,
    dtype *ue2uc_, int ue2uc_r, int ue2uc_r_padded, 
    int ue2uc_c,
    dtype *uc2ue_, int uc2ue_r, int uc2ue_c)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

  if(bid < n_boxes_) {
    bid += reduction_offset;
    int i, j, k;
    /* this should be 8 * uc2ue_r_padded (SP[UE]) */
    /* NP=3 ==> 256 */
    /* NP=4 ==> 512 */
    __shared__ dtype tmpDen[UC2UE_R_PADDED];
    /* size should be 8x ue2uc_r_padded (SP[UC]) */
    /* NP=3 ==> 1024 */
    /* NP=4 ==> 1280 */
    __shared__ dtype tmpTl[UE2UC_R_PADDED];

    /* id of the child node/box of current node/box */
    /* it should be 0~7 as this is an octree */
    int child_id = tid / num_thr_per_child;
    /* thread ID for the child the thread is responsible for */
    /* it should be anywhere from 0~(num_thr_per_child-1) */
    int child_tid = tid % num_thr_per_child;

    /* first, process current node's child's children */
    /* 1. Child 0 goes through each of ITS 8 children and calculate uden.
     * 2. Calculate Child 0's uden 
     * 3. Calculate the current box/node's uden
     */

    /* Identify the real node IDs for the children nodes */
    int cur_child_id = children[bid] + child_id;

    /* each of these children nodes have 8 children of its own and they SHOULD
     * all be leaf nodes
     */
    /* needed variables */
    int ue2uc_index = 0;
    for(i = tid; i < UE2UC_R_PADDED; i += blockDim.x) {
      tmpTl[i] = 0.0;
    }
    __syncthreads ();

    /* now, go through the 8 children */
    for(i = children[cur_child_id]; i < children[cur_child_id] + 8; i++) {
      dtype* den = &src_upw_equ_den_[i * uc2ue_r_padded];
      dtype sclvec = __powf (2.0, (node_depth + 2)); 

      /* scale */	
      for(j = child_tid; j < uc2ue_r_padded; j += num_thr_per_child) {
        tmpDen[child_id * uc2ue_r_padded + j] = den[j] * sclvec;
      }
      __syncthreads();

      dtype *UE2UCii = &ue2uc_[ue2uc_index * ue2uc_r_padded * ue2uc_c];
      /* dgemv */
      for(j = child_tid; j < ue2uc_r_padded; j += num_thr_per_child) {
        dtype tmp = tmpTl[child_id * ue2uc_r_padded + j];
        for(k = 0; k < ue2uc_c; k++) {
          tmp += UE2UCii[k * ue2uc_r_padded + j] * 
            tmpDen[child_id * uc2ue_r_padded + k];
        }
        tmpTl[child_id * ue2uc_r_padded + j] = tmp;
      }
      ue2uc_index++;
    }
    __syncthreads ();	


    /* At this point, each block of num_thr_per_child 
     * has accumulated the results of its 8 leaf nodes
     * Thus, there are 8 sets of tl_pos, each of which belongs to a block
     * Now, we must do dgemv with UC2UE matrix and tl_pos
     */
    dtype sclvec = __powf (2.0, -(node_depth + 1));
    dtype sclvec_ = __powf (2.0, (node_depth + 1));
    dtype tmp;
    for(i = child_tid; i < uc2ue_r_padded; i += num_thr_per_child) {
      tmp = 0.0;
      for(j = 0; j < uc2ue_c; j++) {
        tmp += uc2ue_[j * uc2ue_r_padded + i] * 
          tmpTl[child_id * ue2uc_r_padded + j];
      }
      tmp = tmp * sclvec;

      src_upw_equ_den_[cur_child_id * uc2ue_r_padded + i] = tmp;
      tmpDen[child_id * uc2ue_r_padded + i] = tmp * sclvec_;
    }

    /* Now that we have the uden for all 8 of its children, do the actual
     * computation required for current box/node 
     */
    /* Do each block of num_thr_per_child do dgemv for its own uden and 
     * UE2UC matrix */
    dtype* UE2UCii = &ue2uc_[child_id * ue2uc_r_padded * ue2uc_c];
    for(i = child_tid; i < ue2uc_r_padded; i += num_thr_per_child) {
      dtype tmp_r = 0.0;
      for(j = 0; j < ue2uc_c; j++) {
        tmp_r += UE2UCii[j * ue2uc_r_padded + i] * 
          tmpDen[child_id * uc2ue_r_padded + j];
      }
      tmpTl[child_id * ue2uc_r_padded + i] = tmp_r;
    }
    __syncthreads ();

    /* reduce the 8 results of degmv on tmpTl (8 * ue2uc_r_padded) */
    /* 4 and 4 */
    for(i = tid; i < UE2UC_R_PADDED / 2; i += blockDim.x) {
      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 2)];
    }
    __syncthreads ();
    /* 2 and 2 */
    for(i = tid; i < UE2UC_R_PADDED / 4; i += blockDim.x) {
      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 4)];
    }
    __syncthreads ();
    /* 1 and 1 */
    for(i = tid; i < UE2UC_R_PADDED / 8; i += blockDim.x) {
      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 8)];
    }
    __syncthreads ();


    /* Finally do dgemv with UC2UE matrix */
    sclvec = __powf (2.0, -node_depth);
    for(i = tid; i < uc2ue_r_padded; i += blockDim.x) {
      tmp = 0.0;
      for(j = 0; j < uc2ue_c; j++) {
        tmp += uc2ue_[j * uc2ue_r_padded + i] * tmpTl[j];
      }

      tmp = tmp * sclvec;

      src_upw_equ_den_[bid * uc2ue_r_padded + i] = tmp;
    }	
  }
}

/* ------------------------------------------------------------------------
 */

__global__
  void
up_eval__gpu_reduction_last (int num_thr_per_child, int n_boxes_, 
    int reduction_offset, int node_depth,
    int *children, 
    dtype *src_upw_equ_den_, int uc2ue_r_padded,
    dtype *ue2uc_, int ue2uc_r, int ue2uc_r_padded, 
    int ue2uc_c,
    dtype *uc2ue_, int uc2ue_r, int uc2ue_c)

{
  int i, j;

  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

  if(bid < n_boxes_) {
    bid += reduction_offset;

    __shared__ dtype tmpDen[UC2UE_R_PADDED];
    __shared__ dtype tmpTl[UE2UC_R_PADDED];

    /* 0 ~ 7 */
    int child_id = tid / num_thr_per_child;
    /* 0 ~ num_thr_per_child */
    int child_tid = tid % num_thr_per_child;

    /* node ID of one of 8 children */
    int cur_child_id = children[bid] + child_id;

    /* src_upw_equ_den for the child */
    dtype *den = &src_upw_equ_den_[cur_child_id * uc2ue_r_padded];

    /* scale */
    dtype sclvec = __powf (2.0, (node_depth + 1));
    dtype tmp; 
    for(i = child_tid; i < uc2ue_r_padded; i+= num_thr_per_child) {
      tmp = den[i];
      tmpDen[child_id * uc2ue_r_padded + i] = tmp * sclvec;
    }
    __syncthreads ();

    /* dgemv */
    dtype *UE2UCii = &ue2uc_[child_id * ue2uc_r_padded * ue2uc_c];
    for(i = child_tid; i < ue2uc_r_padded; i += num_thr_per_child) {
      dtype tmp_r = 0.0;
      for(j = 0; j < ue2uc_c; j++) {
        tmp_r += UE2UCii[j * ue2uc_r_padded + i] *
          tmpDen[child_id * uc2ue_r_padded + j];
      }
      tmpTl[child_id * ue2uc_r_padded + i] = tmp_r;
    }
    __syncthreads ();

    /* reduce the 8 children */
    /* 4 and 4 */
    for(i = tid; i < UE2UC_R_PADDED / 2; i += blockDim.x) {
      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 2)];
    }
    __syncthreads ();
    /* 2 and 2 */
    for(i = tid; i < UE2UC_R_PADDED / 4; i += blockDim.x) {
      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 4)];
    }
    __syncthreads ();
    /* 1 and 1 */
    for(i = tid; i < UE2UC_R_PADDED / 8; i += blockDim.x) {
      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 8)];
    }
    __syncthreads ();

    /* final dgemv and scale */
    sclvec = __powf (2.0, -node_depth);
    for(i = tid; i < uc2ue_r_padded; i += blockDim.x) {
      tmp = 0.0;
      for(j = 0; j < uc2ue_c; j++) {
        tmp += uc2ue_[j * uc2ue_r_padded + i] * tmpTl[j];
      }

      tmp = tmp * sclvec;

      src_upw_equ_den_[bid * uc2ue_r_padded + i] = tmp;
    }	
  }
}

__global__
void
up_eval__gpu_non_leaf (int num_thr_per_child, 	
											 int n_boxes_,
											 int node_depth,
											 int *children,
											 dtype *src_upw_equ_den_,
											 int uc2ue_r, int uc2ue_c,
											 int uc2ue_r_padded,
											 dtype *uc2ue_,
											 int ue2uc_r, int ue2uc_c,
											 int ue2uc_r_padded,
											 dtype *ue2uc_,
											 int *tag_,
											 int *depth_)
{
  int i, j;

  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

	__shared__ dtype tmpDen[UC2UE_R_PADDED];
	__shared__ dtype tmpTl[UE2UC_R_PADDED];

	if(bid < n_boxes_) {
		if(depth_[bid] == node_depth && children[bid] != -1) {
	    /* 0 ~ 7 */
	    int child_id = tid / num_thr_per_child;

	    /* 0 ~ num_thr_per_child */
	    int child_tid = tid % num_thr_per_child;

	    /* node ID of one of 8 children */
	    int cur_child_id = children[bid] + child_id;

			dtype *den = &src_upw_equ_den_[cur_child_id * uc2ue_r_padded];

		  dtype sclvec = __powf (2.0, (node_depth + 1));
			dtype tmp;


			if(tag_[cur_child_id] & LET_SRCNODE) {
				/* scale */
		    for(i = child_tid; i < uc2ue_r_padded; i+= num_thr_per_child) {
		      tmp = den[i];
		      tmpDen[child_id * uc2ue_r_padded + i] = tmp * sclvec;
		    }

				/* dgemv */
		    dtype *UE2UCii = &ue2uc_[child_id * ue2uc_r_padded * ue2uc_c];
		    for(i = child_tid; i < ue2uc_r_padded; i += num_thr_per_child) {
		      dtype tmp_r = 0.0;
		      for(j = 0; j < ue2uc_c; j++) {
		        tmp_r += UE2UCii[j * ue2uc_r_padded + i] *
		          			 tmpDen[child_id * uc2ue_r_padded + j];
		      }
		      tmpTl[child_id * ue2uc_r_padded + i] = tmp_r;
		    }
			} else {
				for(i = child_tid; i < ue2uc_r_padded; i += num_thr_per_child) {
					tmpTl[child_id * ue2uc_r_padded + i] = 0.0;
				}
			} /* child == SRC */
			__syncthreads ();


	    /* reduce the 8 children */
	    /* 4 and 4 */
	    for(i = tid; i < UE2UC_R_PADDED / 2; i += blockDim.x) {
	      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 2)];
	    }
	    __syncthreads ();
	    /* 2 and 2 */
	    for(i = tid; i < UE2UC_R_PADDED / 4; i += blockDim.x) {
	      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 4)];
	    }
	    __syncthreads ();
	    /* 1 and 1 */
	    for(i = tid; i < UE2UC_R_PADDED / 8; i += blockDim.x) {
	      tmpTl[i] += tmpTl[i + (UE2UC_R_PADDED / 8)];
	    }
	    __syncthreads ();


	    /* final dgemv and scale */
	    sclvec = __powf (2.0, -node_depth);
	    for(i = tid; i < uc2ue_r; i += blockDim.x) {
	      tmp = 0.0;
	      for(j = 0; j < uc2ue_c; j++) {
	        tmp += uc2ue_[j * uc2ue_r_padded + i] * tmpTl[j];
	      }

	      tmp = tmp * sclvec;

	      src_upw_equ_den_[bid * uc2ue_r_padded + i] = tmp;
	    }
		} /* depth == node_depth */
	} /* bid < n_boxes_ */
}

/* ------------------------------------------------------------------------
 */
void
up_calc__gpu_reduction(FMMWrapper_t *f)
{
	int i;

  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	const int NB = get_thread_block_size_up_reduce ();
	const int NG = nodeVec.size ();
	const int num_thr_per_child = NB / 8;

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);


	/* go through each depth and do up_calc */
	for(i = f->tree_max_depth - 1; i >= 0; i--) {
		up_eval__gpu_non_leaf <<<GB, TB>>> (num_thr_per_child,
																				nodeVec.size (),
																				i, 
																				f->child_d_,
																				f->SRC_UPW_EQU_DEN_d_,
																				f->UC2UE_r, f->UC2UE_c,
																				f->UC2UE_r_padded,
																				f->UC2UE_d_,
																				f->UE2UC_r, f->UE2UC_c,
																				f->UE2UC_r_padded,
																				f->UE2UC_d_,
																				f->tag_d_,
																				f->depth_d_);
	}
	hipDeviceSynchronize ();
	gpu_check_error (stderr);
}

/* ------------------------------------------------------------------------
 */
  int
up_calc__gpu (FMMWrapper_t* f)
{

  Boxes__gpu__t* S;
  assert (f);
  S = &f->S_d_;

  const int NB = get_thread_block_size_up ();
  const int NG = S->n_boxes_;

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

  up_eval__gpu <<<GB, TB>>> (S->n_boxes_, S->Bptr_, S->Bn_, 
      S->x_, S->y_, S->z_, S->w_,
      f->radius_d_, 
      f->center0_d_, f->center1_d_, f->center2_d_,
      f->SP_UC_size, f->SP_UC_size_padded, f->SP_UC_d_,
      f->UC2UE_r, f->UC2UE_r_padded, f->UC2UE_c, 
      f->UC2UE_d_,
      /* f->num_non_leaf_nodes, */ f->depth_d_,
      f->SRC_UPW_EQU_DEN_d_);

  hipDeviceSynchronize ();
  gpu_check_error (stderr);

  up_calc__gpu_reduction (f);
  return 0;
}

/* ------------------------------------------------------------------------
 */
/* ULIST FUNCTIONS */
__global__
  void
ulist_eval__gpu (int n_boxes__trg, int *Bptr__trg, int *Bn__trg, 
    dtype *x__trg, dtype *y__trg, dtype *z__trg, dtype *w__trg,
    int n_boxes__src, int *Bptr__src, int *Bn__src,
    dtype *x__src, dtype *y__src, dtype *z__src, dtype *w__src,
    int* Ptr__u, int* L__u)
{
  /* thread and block ID's */
  const int tid = threadIdx.x;
  const int bid = blockIdx.y * gridDim.x + blockIdx.x;


  if(bid < n_boxes__trg) {

    int i, j, k;

    /* points to beginning and end of this thread block's (target's) ulist 
       neighbor */
    const int u_begin = Ptr__u[bid];
    const int u_end = Ptr__u[bid + 1];

    /* points to first and last point in this thread block (target) */
    const int trg_begin = Bptr__trg[bid];
    const int trg_end = Bptr__trg[bid] + Bn__trg[bid];

    /* Loop over each target point */
    for(i = trg_begin + tid; i < trg_end; i += blockDim.x) {
      dtype xt = x__trg[i];
      dtype yt = y__trg[i];
      dtype zt = z__trg[i];
      dtype wt = 0.0;

      /* For each target point, loop over the soure boxes in the ulist */
      for(j = u_begin; j < u_end; j++) {
        const int src_id = L__u[j];
        const int src_begin = Bptr__src[src_id];
        const int src_end = Bptr__src[src_id] + Bn__src[src_id];

        /* Loop over points in each source box */
        for(k = src_begin; k < src_end; k++) {
          dtype xs = x__src[k];
          dtype ys = y__src[k];
          dtype zs = z__src[k];
          dtype ws = w__src[k];

          xs = xt - xs;
          ys = yt - ys;
          zs = zt - zs;

          dtype rsq = xs * xs + ys * ys + zs * zs;
          rsq = rsqrt (rsq);

          wt += ws * rsq;
        }
      }
      w__trg[i] = OOFP_R * wt;
    }
  }
}

/* ------------------------------------------------------------------------
 */
  int
ulist_calc__gpu (FMMWrapper_t* f)
{
  /* Source and target boxes on GPU */
  const Boxes__gpu__t* S;
  Boxes__gpu__t* T;

  /* Ulist on GPU */
  const UList__gpu__t* U;

  assert (f);

  S = &f->S_d_;
  T = &f->T_d_;
  U = &f->U_d_;

  const int NB = get_thread_block_size_ulist ();
  const int NG = T->n_boxes_;
  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

  ulist_eval__gpu <<<GB, TB>>> (T->n_boxes_, T->Bptr_, T->Bn_, 
      T->x_, T->y_, T->z_, T->w_,
      S->n_boxes_, S->Bptr_, S->Bn_, 
      S->x_, S->y_, S->z_, S->w_,
      U->Ptr_, U->L_);

  // hipDeviceSynchronize ();
  gpu_check_error (stderr);


  return 0;
}
/* ------------------------------------------------------------------------
 */
/* ------------------------------------------------------------------------
 */
/* VLIST FUNCTIONS */
#if 0
/* size should be RP->n =(2np)^3 */
/* NP=3 ==> 224 */
/* NP=4 ==> 896 */
//#define RP_N 216
#define RP_N 512
//#define RP_N 896 

/* incorrectly named */
/* UC2UE_R_PADDED (fake) <= 8 * UC2UE_R_PADDED (real) */
#define UC2UE_R (UC2UE_R_PADDED/8)
#endif

__global__
  void
compute_fft_src__gpu_eval(int n_boxes_, int *depth, int np,
    dtype *src_upw_equ_den_, int uc2ue_r_padded,
    dtype *reg_den_, int reg_den_size)

{
  __shared__ dtype reg_den[RP_N];
  __shared__ dtype tmp_src[UC2UE_R];
  __shared__ int tmp_index[NP_CUBED_POWER_OF_2];
  const int tid = threadIdx.x;
  const int bid = blockIdx.y * gridDim.x + blockIdx.x;

  if(bid < n_boxes_) {
    int i, j, k;
    /* initialize reg_den 
     * this is needed since all of it will be written back, and those
     * that doesn't have src_upw_equ_den written to it should be 0.0
     */
    for(i = tid; i < reg_den_size; i += blockDim.x) {
      reg_den[i] = 0.0;
    }	
    /* sync not needed */


    int l = (int) __powf (2.0, depth[bid]); /* powf is okay */
    int rgnum = 2 * np;
    int cnt;
    int index;

    dtype *src = &src_upw_equ_den_[bid * uc2ue_r_padded];
    /* load src_upw_equ_den */
    for(i = tid; i < uc2ue_r_padded; i += blockDim.x) {
      tmp_src[i] = src[i];
    }
    __syncthreads ();

    /* compute index for each thread */
    cnt = 0;
    index = 0;
    if(tid == 0) {
      for(i = 0; i < np ; i++) {
        for(j = 0; j < np; j++) {
          for(k = 0; k < np; k++) {
            if(i == 0 || i == np - 1 || j == 0 || j == np - 1 
                || k == 0 || k == np - 1) {
              tmp_index[index] = cnt;
              cnt++;
              /* index is the thread id and cnt is the index into the source
                 array that the thread will be accessing */
            }
            index++;
          }
        }
      }
    }
    __syncthreads ();

    dtype tmp;

    i = tid / (np * np);
    j = (tid % (np * np)) / np;
    k = tid % np;

    /* np^3 - (np-2)^3 threads will be valid */
    if(tid < (np * np * np)) {
      if(i == 0 || i == np - 1 || j == 0 || j == np - 1 
          || k == 0 || k == np - 1) {
        tmp = tmp_src[tmp_index[tid]] * l;

        int rgoff = (k + np / 2) * rgnum * rgnum + ( j + np / 2) * rgnum + 
          (i + np / 2);
        reg_den[rgoff] = tmp;
      }
    }
    __syncthreads ();

    for(i = tid; i < reg_den_size ; i+= blockDim.x) {
      reg_den_[bid * reg_den_size + i] = reg_den[i];
    }
  }
}

  void
compute_fft_src__gpu (FMMWrapper_t *f, AllNodes *All_N)
{
  vector<NodeTree>& nodeVec = *All_N->N;
  const int np = getenv__accuracy ();

  /* source: All_N->src_upw_equ_den[i * pln_size (UE, SP)]; padded, SP[UE] */
  /* result: All_N->eff_den[i * eff_size]; padded, (2+2np)*(2np)*(2np) */

  /* scale src_upw_equ_den into tmp_den */
  /* exec samDen2RegDen (tmp_den => reg_den) 
   * this just expands tmp_den into a larger reg_den array 
   */
  /* store the expanded array in shared memory to do coalesced writes */
  int NB; 
  if(f->UC2UE_r_padded <= 32)
    NB = 32;
  else if(f->UC2UE_r_padded <= 64)
    NB = 64;
  else if(f->UC2UE_r_padded <= 128)
    NB = 128;
  else if(f->UC2UE_r_padded <= 256)
    NB = 256;
  else if(f->UC2UE_r_padded <= 512)
    NB = 512;
  else {
    NB = 1024;
    printf("compute_fft_src__gpu: THIS MIGHT CAUSE A PROBLEM\n");
  }

  const int NG = nodeVec.size ();
  dim3 GB(65535, (NG / 65535) + 1, 1);
  dim3 TB(NB, 1, 1);

  compute_fft_src__gpu_eval <<<GB, TB>>> (nodeVec.size (), 
      f->depth_d_,
      getenv__accuracy (),
      f->SRC_UPW_EQU_DEN_d_, 	
      f->UC2UE_r_padded,
      f->reg_den_d_, f->reg_den_size);

  hipDeviceSynchronize ();
  gpu_check_error (stderr);

	#if MIN_DATA
		hipFree (f->SRC_UPW_EQU_DEN_d_);
	#endif

	hipfftResult ccc;
  hipfftHandle plan;
  int nnn[3]; nnn[0] = np * 2; nnn[1] = np * 2; nnn[2] = np * 2;
  hipfftPlanMany (&plan, 3, nnn, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, 
      nodeVec.size ());
  ccc = hipfftExecD2Z (plan, f->reg_den_d_, (hipfftDoubleComplex *) f->vlist_src_d_);

  if(ccc != 0) {
    printf("AAA: %d\n", ccc);
    printf("SUCCESS: %d\n", HIPFFT_SUCCESS);
    printf("INVALID PLAN: %d\n", HIPFFT_INVALID_PLAN);
    printf("ALLOC FAILED: %d\n", HIPFFT_ALLOC_FAILED);
    printf("INVALID TYPE: %d\n", HIPFFT_INVALID_TYPE);
    printf("INVALID VALUE: %d\n", HIPFFT_INVALID_VALUE);
    printf("INTERNAL ERROR: %d\n", HIPFFT_INTERNAL_ERROR);
    printf("EXEC FAILED: %d\n", HIPFFT_EXEC_FAILED);
    printf("SETUP FAILED: %d\n", HIPFFT_SETUP_FAILED);
    printf("INVALID SIZE: %d\n", HIPFFT_INVALID_SIZE);
  }

	hipfftDestroy (plan);

}


  __global__
void compute_fft_trans__gpu_eval (int rp_n_, int np, dtype* tt)
{
  int tid = threadIdx.x;
  // int bid = blockIdx.y * gridDim.x + blockIdx.x;
  int i1 = (blockIdx.x % 7) - 3;
  int i2 = (blockIdx.x / 7) - 3;
  int i3 = blockIdx.y - 3;

  int idx;
  dtype R = 1.0;
  dtype step = 2.0 / (np - 1);

  dtype denPosX, denPosY, denPosZ;

  if(abs (i1) > 1 || abs (i2) > 1 || abs (i3) > 1) {
    idx = (i1 + 3) + (i2 + 3) * 7 + (i3 + 3) * 7 * 7;
    denPosX = (dtype) i1 * 2.0 * R;
    denPosY = (dtype) i2 * 2.0 * R;
    denPosZ = (dtype) i3 * 2.0 * R;

    /* daxpy can be skipped because a = 1.0 and y is 0.0 */
    /* chkPosX/Y/Z is same as RP.x/y/z */
    for(int i = tid; i < rp_n_; i += blockDim.x) {
      /* compute RP.x, RP.y, RP.z */
      int ii = i % (2 * np);
      int jj = (i / (2 * np)) % (2 * np);
      int kk = i / (2 * np * 2 * np);

      int gi = (ii < np) ? ii : (ii - 2 * np);
      int gj = (jj < np) ? jj : (jj - 2 * np);
      int gk = (kk < np) ? kk : (kk - 2 * np);

      dtype x = R * gi * step;
      dtype y = R * gj * step;
      dtype z = R * gk * step;


      /* kernel */
      x = denPosX - x;
      y = denPosY - y;
      z = denPosZ - z;
      dtype r = x * x + y * y + z * z;
      r = rsqrt (r);
      tt[idx * rp_n_ + i] = OOFP_R * r;

      // if(idx == 0) printf("%d %f\n", i, OOFP_R * r);
    }
  }

}

  void
compute_fft_trans__gpu (FMMWrapper_t *f, AllNodes *All_N)
{
  const int NB = get_thread_block_size_fft_trans ();
  const int np = getenv__accuracy ();

  dim3 GB ((7*7), 7, 1);
  dim3 TB (NB, 1, 1);

  compute_fft_trans__gpu_eval <<<GB, TB>>> (f->RP_n_, 
      np, 
      f->tt);	
  hipDeviceSynchronize ();
  gpu_check_error (stderr);

	hipfftResult ccc;
  hipfftHandle plan;
  int nnn[3]; nnn[0] = 2 * np; nnn[1] = 2 * np; nnn[2] = 2 * np;
  hipfftPlanMany (&plan, 3, nnn, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, (7 * 7 * 7));
  ccc = hipfftExecD2Z (plan, f->tt, (hipfftDoubleComplex *) f->vlist_trans_d_);

  if(ccc != 0) {
    printf("AAA: %d\n", ccc);
    printf("SUCCESS: %d\n", HIPFFT_SUCCESS);
    printf("INVALID PLAN: %d\n", HIPFFT_INVALID_PLAN);
    printf("ALLOC FAILED: %d\n", HIPFFT_ALLOC_FAILED);
    printf("INVALID TYPE: %d\n", HIPFFT_INVALID_TYPE);
    printf("INVALID VALUE: %d\n", HIPFFT_INVALID_VALUE);
    printf("INTERNAL ERROR: %d\n", HIPFFT_INTERNAL_ERROR);
    printf("EXEC FAILED: %d\n", HIPFFT_EXEC_FAILED);
    printf("SETUP FAILED: %d\n", HIPFFT_SETUP_FAILED);
    printf("INVALID SIZE: %d\n", HIPFFT_INVALID_SIZE);
  }
	
	hipfftDestroy (plan);


}

__global__
  void
compute_ifft_trg__gpu_eval_scale_nrmfc (int n_boxes_, 
    dtype *trg_,
    int vlist_array_size,
    int rp_n_)
{
  int i;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

  if(bid < n_boxes_) {
    dtype nrmfc = 1.0 / (dtype) rp_n_;
    for(i = threadIdx.x; i < vlist_array_size; i += blockDim.x) {
      trg_[bid * vlist_array_size + i] = trg_[bid * vlist_array_size + i] * 
        nrmfc;
    }
  }
}

#if 0
/* this should be qual to sp_dc_n_padded */
/* NP=3 ==> 32 */
//#define SP_DC_N 32
#define SP_DC_N 64
#endif

__global__
  void
compute_ifft_trg__gpu_regVal2SamVal(int n_boxes_, int np,
    dtype *reg_den_, int reg_den_size_,
    dtype *trg_dwn_chk_val_, int sp_dc_n_padded)
{
  int bid = blockIdx.y * gridDim.x + blockIdx.x;
  int tid = threadIdx.x;

  __shared__ int tmp_index[NP_CUBED_POWER_OF_2];
  __shared__ dtype tmp_trg[SP_DC_N];

  if(bid < n_boxes_) {
    int i, j, k;

    /* initialize tmp_trg 
     * this is needed since all of it will be written back, and those
     * that doesn't have reg_den_ written to it should be 0.0
     */
    for(i = tid; i < sp_dc_n_padded; i += blockDim.x) {
      tmp_trg[i] = 0.0;
    }
    __syncthreads ();

    int rgnum = 2 * np;

    int cnt = 0;
    int index = 0;
    if(tid == 0) {
      for(i = 0; i < np ; i++) {
        for(j = 0; j < np; j++) {
          for(k = 0; k < np; k++) {
            if(i == 0 || i == np - 1 || j == 0 || j == np - 1
                || k == 0 || k == np - 1) {
              /* 'cnt' is index into the SP[DC].n sized array 'tmp_trg' for 
               * thread 'index' */
              tmp_index[index] = cnt;
              cnt++;
            }
            index++;
          }
        }
      }
    }
    __syncthreads ();

    i = tid / (np * np);
    j = (tid % (np * np)) / np;
    k = tid % np;

    if(tid < (np * np * np)) {
      if(i == 0 || i == np - 1 || j == 0 || j == np - 1
          || k == 0 || k == np - 1) {
        int rgoff = (k + np / 2) * rgnum * rgnum + (j + np / 2) * rgnum + 
          (i + np / 2);
        tmp_trg[tmp_index[tid]] = reg_den_[bid * reg_den_size_ + rgoff];
      }
    }
    __syncthreads ();

    for(i = tid; i < sp_dc_n_padded; i += blockDim.x) {
      trg_dwn_chk_val_[bid * sp_dc_n_padded + i] = tmp_trg[i];
      //if(bid == 0) printf("%d %f\n", i, tmp_trg[i]);
    }
  }
}

  void
compute_ifft_trg__gpu (FMMWrapper_t *f, AllNodes *All_N)
{
  /* ============================================================ */
  /* source: eff_val[288] */
  /* target: trg_dwn_chk_val[26] */
  /* ============================================================ */
  /* scale source nrmfc */
  vector<NodeTree>& nodeVec = *All_N->N;

  const int NG = nodeVec.size ();
  const int NB = get_thread_block_size_vlist ();

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

  compute_ifft_trg__gpu_eval_scale_nrmfc <<<GB, TB>>> (nodeVec.size (),
      f->vlist_trg_d_,
      f->vlist_array_size,
      f->RP_n_);

  hipDeviceSynchronize ();
  gpu_check_error (stderr);


  /* ifft */
  hipfftResult ccc;
  const int np = getenv__accuracy ();
  hipfftHandle plan;
  int nnn[3]; nnn[0] = 2 * np; nnn[1] = 2 * np; nnn[2] = 2 * np;


  #if 0
  printf("doing FFT in multiple steps\n");
  int fft_size = 1024;
  int num_fft_iter = (nodeVec.size () + fft_size - 1) / fft_size;

  hipfftPlanMany (&plan, 3, nnn, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D, fft_size);

  int i;
  for(i = 0; i < num_fft_iter - 1; i++) {
		ccc = hipfftExecZ2D (plan, (hipfftDoubleComplex *) &f->vlist_trg_d_[i * fft_size * f->vlist_array_size], &f->reg_den_d_[i * fft_size * f->reg_den_size]);


    if(ccc != 0) {
     printf("CCC: %d\n", ccc);
     printf("SUCCESS: %d\n", HIPFFT_SUCCESS);
     printf("INVALID PLAN: %d\n", HIPFFT_INVALID_PLAN);
     printf("ALLOC FAILED: %d\n", HIPFFT_ALLOC_FAILED);
     printf("INVALID TYPE: %d\n", HIPFFT_INVALID_TYPE);
     printf("INVALID VALUE: %d\n", HIPFFT_INVALID_VALUE);
     printf("INTERNAL ERROR: %d\n", HIPFFT_INTERNAL_ERROR);
     printf("EXEC FAILED: %d\n", HIPFFT_EXEC_FAILED);
     printf("SETUP FAILED: %d\n", HIPFFT_SETUP_FAILED);
     printf("INVALID SIZE: %d\n", HIPFFT_INVALID_SIZE);
   }
  }

  if(nodeVec.size () % fft_size != 0) {
    hipfftDestroy (plan);
    hipfftPlanMany (&plan, 3, nnn, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,
                   (nodeVec.size () % fft_size ));
  }

	ccc = hipfftExecZ2D (plan, (hipfftDoubleComplex *) &f->vlist_trg_d_[i * fft_size * f->vlist_array_size], &f->reg_den_d_[i * fft_size * f->reg_den_size]);

  if(ccc != 0) {
    printf("CCC: %d\n", ccc);    printf("SUCCESS: %d\n", HIPFFT_SUCCESS);
    printf("INVALID PLAN: %d\n", HIPFFT_INVALID_PLAN);
    printf("ALLOC FAILED: %d\n", HIPFFT_ALLOC_FAILED);
    printf("INVALID TYPE: %d\n", HIPFFT_INVALID_TYPE);
    printf("INVALID VALUE: %d\n", HIPFFT_INVALID_VALUE);
    printf("INTERNAL ERROR: %d\n", HIPFFT_INTERNAL_ERROR);
    printf("EXEC FAILED: %d\n", HIPFFT_EXEC_FAILED);
    printf("SETUP FAILED: %d\n", HIPFFT_SETUP_FAILED);
    printf("INVALID SIZE: %d\n", HIPFFT_INVALID_SIZE);
  }
  #endif



	#if 1
  hipfftPlanMany (&plan, 3, nnn, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D, 
      nodeVec.size ());
  ccc = hipfftExecZ2D (plan, (hipfftDoubleComplex *) f->vlist_trg_d_, f->reg_den_d_);

  if(ccc != 0) {
    printf("CCC: %d\n", ccc);
    printf("SUCCESS: %d\n", HIPFFT_SUCCESS);
    printf("INVALID PLAN: %d\n", HIPFFT_INVALID_PLAN);
    printf("ALLOC FAILED: %d\n", HIPFFT_ALLOC_FAILED);
    printf("INVALID TYPE: %d\n", HIPFFT_INVALID_TYPE);
    printf("INVALID VALUE: %d\n", HIPFFT_INVALID_VALUE);
    printf("INTERNAL ERROR: %d\n", HIPFFT_INTERNAL_ERROR);
    printf("EXEC FAILED: %d\n", HIPFFT_EXEC_FAILED);
    printf("SETUP FAILED: %d\n", HIPFFT_SETUP_FAILED);
    printf("INVALID SIZE: %d\n", HIPFFT_INVALID_SIZE);
  }
	#endif

	hipfftDestroy(plan);

	#if MIN_DATA
		alloc__TRG_DWN_CHK_VAL__ (f);
	#endif


  /* regVal2SamVal */
  int NB_;
  if(f->SP_DC_n_padded_ <= 32)
    NB_ = 32;
  else if(f->SP_DC_n_padded_ <= 64)
    NB_ = 64;
  else if(f->SP_DC_n_padded_ <= 128)
    NB_ = 128;
  else if(f->SP_DC_n_padded_ <= 256)
    NB_ = 256;
  else if(f->SP_DC_n_padded_ <= 512)
    NB_ = 512;
  else {
    NB_ = 1024;
    printf("compute_ifft_src__gpu: THIS MIGHT CAUSE A PROBLEM\n");
  }

  dim3 GB_ (65535, (NG / 65535) + 1, 1);
  dim3 TB_ (NB_, 1, 1);


  compute_ifft_trg__gpu_regVal2SamVal <<<GB_, TB_>>> (nodeVec.size (), np,
      // f->reg_den_ifft_d_,
      f->reg_den_d_,
      f->reg_den_size,
      f->TRG_DWN_CHK_VAL_d_,
      f->SP_DC_n_padded_);

  hipDeviceSynchronize ();
  gpu_check_error (stderr);

}

#define VLIST_SIZE 288

__global__
  void
vlist_calc__gpu_eval (int n_boxes_, dtype *src_, dtype *trans_, dtype *trg_,
    int *vlist_, int* tlist_, int* list_ptr_, 
    int vlist_array_size)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;


  if(bid < n_boxes_) {
    int i, j;
    int vi, ti;

    dtype re, im;
    dtype tmp_src_re, tmp_src_im;
    dtype tmp_trans_re, tmp_trans_im;

    for(i = tid; i < (vlist_array_size / 2); i+= blockDim.x) {
      re = 0.0; im = 0.0;
      for(j = list_ptr_[bid]; j < list_ptr_[bid + 1]; j++) {
        vi = vlist_[j];
        ti = tlist_[j];

        tmp_src_re = src_[vi * vlist_array_size + i * 2 + 0];
        tmp_src_im = src_[vi * vlist_array_size + i * 2 + 1];

        tmp_trans_re = trans_[ti * vlist_array_size + i * 2 + 0];
        tmp_trans_im = trans_[ti * vlist_array_size + i * 2 + 1];


        re += tmp_src_re * tmp_trans_re - tmp_src_im * tmp_trans_im;
        im += tmp_src_re * tmp_trans_im + tmp_src_im * tmp_trans_re;
      }

      trg_[bid * vlist_array_size + i * 2 + 0] = re;
      trg_[bid * vlist_array_size + i * 2 + 1] = im;

    }
  }
}

  void
vlist_calc__gpu_ (FMMWrapper_t *f, AllNodes *All_N)
{
  vector<NodeTree>& nodeVec = *All_N->N;
  const int NB = get_thread_block_size_vlist ();
  const int NG = nodeVec. size();

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

  vlist_calc__gpu_eval <<<GB, TB>>> (nodeVec.size (), f->vlist_src_d_, 
      f->vlist_trans_d_, f->vlist_trg_d_,
      f->vlist_d_, f->tlist_d_, f->vlist_ptr_d_,
      f->vlist_array_size);

  hipDeviceSynchronize ();
  gpu_check_error (stderr);

}

/* ------------------------------------------------------------------------
 */

int
vlist_calc__gpu (FMMWrapper_t *f)
{
  AllNodes *All_N = f->AN;
  compute_fft_src__gpu (f, All_N);
  compute_fft_trans__gpu (f, All_N);
  vlist_calc__gpu_ (f, All_N);
  compute_ifft_trg__gpu (f, All_N);

  return 0;
}
/* ------------------------------------------------------------------------
 */

/* ------------------------------------------------------------------------
 */
#if 0
/* DOWN_CALC FUNCTIONS */
/* NP=3 ==> 256 */
/* NP=4 ==> 512 */
//#define SP_DE_N_PADDED_8 256
#define SP_DE_N_PADDED_8 512
#endif
__global__
  void
down_eval__gpu(int num_thr_per_child, int n_boxes_, int offset, int* children,
    int3 *path2Node, int max_depth,
    dtype *trg_dwn_chk_val_, int sp_dc_n, int sp_dc_n_padded,
    dtype *trg_dwn_equ_den_, int sp_de_n, int sp_de_n_padded,
    dtype *dc2de_, int dc2de_r, int dc2de_r_padded, int dc2de_c,
    dtype *de2dc_, int de2dc_r, int de2dc_r_padded, int de2dc_c)
{
  /* at minimum size of 8x SP[DE].n padded */
  __shared__ dtype tmpDen[SP_DE_N_PADDED_8];
  /* at minimum size of 8x SP[DE].n padded */
  __shared__ dtype tmpDen_[SP_DE_N_PADDED_8];
  __shared__ dtype tmpDen__[SP_DE_N_PADDED_8];

  int i, j, k;	
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

  if(bid < n_boxes_) {
    bid += offset;
    /* ---------------------------------------------------------- */
    /* do nodes at depth = 2 */
    int node_depth = 2;
    /* dgemv */
    for(i = tid; i < dc2de_r_padded; i += blockDim.x) {
      dtype tmp = 0.0;
      for(j = 0; j < dc2de_c; j++) {
        tmp += dc2de_[j * dc2de_r_padded + i] * 
          trg_dwn_chk_val_[bid * sp_dc_n_padded + j];
      }
      tmpDen[i] = tmp;
    }

    /* scale */	
    dtype sclvec = __powf (2.0, -node_depth);
    for(i = tid; i < sp_de_n_padded; i += blockDim.x) {
      tmpDen[i] = tmpDen[i] * sclvec;
#if 0
      trg_dwn_equ_den_[bid * sp_de_n_padded + i] = tmpDen[i];	
#endif
    }
    __syncthreads ();
    /* ---------------------------------------------------------- */

    /* ---------------------------------------------------------- */
    /* do depth 3 */
    /* ID of this node's child at depth 3 */
    int child_id = tid / num_thr_per_child;
    /* sub-tid 0~num_thr_per_child for this child child_id */
    int child_tid = tid % num_thr_per_child;
    /* real child ID */
    int cur_child_id = children[bid] + child_id;

    /* parent's dden/trg_dwn_equ_den is in tmpDen */
    /* scale */
    sclvec = __powf (2.0, node_depth);
    for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
      tmpDen_[child_id * sp_de_n_padded + i] = tmpDen[i] * sclvec;
    }

    int3 cidx_parent = path2Node[bid];
    int3 cidx_child = path2Node[cur_child_id];
    int de2dc_index = (cidx_child.z - 2 * cidx_parent.z) + 
      (cidx_child.y - 2 * cidx_parent.y) * 2 +
      (cidx_child.x - 2 * cidx_parent.x) * 4;
    dtype *de2dc_cur = &de2dc_[de2dc_index * de2dc_r_padded * de2dc_c];
    /* dgemv */
    for(i = child_tid; i < de2dc_r_padded; i += num_thr_per_child) {
      dtype tmp = trg_dwn_chk_val_[cur_child_id * sp_dc_n_padded + i];
      for(j = 0; j < de2dc_c; j++) {
        tmp += de2dc_cur[j * de2dc_r_padded + i] * 
          tmpDen_[child_id * sp_de_n_padded + j];
      }
      tmpDen[child_id * de2dc_r_padded + i] = tmp;
    }

    /* dgemv */
    for(i = child_tid; i < dc2de_r_padded; i += num_thr_per_child) {
      dtype tmp = 0.0;
      for(j = 0; j < dc2de_c; j++) {
        tmp += dc2de_[j * dc2de_r_padded + i] * 
          tmpDen[child_id * de2dc_r_padded + j];
      }
      tmpDen_[child_id * dc2de_r_padded + i] = tmp;
    }
    /* scale */
    node_depth++;
    sclvec = __powf (2.0, -node_depth);
    for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
      tmpDen_[child_id * sp_de_n_padded + i] = 
        tmpDen_[child_id * sp_de_n_padded + i] * sclvec;
    }
    /* ---------------------------------------------------------- */

    if(node_depth == max_depth) {
      /* write back the results to main memory for leaf node computation */
      for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
        trg_dwn_equ_den_[cur_child_id * sp_de_n_padded + i] = 
          tmpDen_[child_id * sp_de_n_padded + i];
      }
    } else {
#if 0
      for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
        trg_dwn_equ_den_[cur_child_id * sp_de_n_padded + i] = 
          tmpDen_[child_id * sp_de_n_padded + i];
      }
#endif
      /* continue down to next level (depth 4) */
      /* each set of num_thr_per_child is responsible for all 8 of its children 
       */
      int first_child = children[cur_child_id];
      /* scale */
      sclvec = __powf (2.0, node_depth);
      for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
        tmpDen_[child_id * sp_de_n_padded + i] =  
          tmpDen_[child_id * sp_de_n_padded + i] * sclvec;
      }
      node_depth++;

      /* go through each child */
      for(i = first_child ; i < first_child + 8; i++) {
        cidx_parent = path2Node[cur_child_id];
        cidx_child = path2Node[i];
        de2dc_index = (cidx_child.z - 2 * cidx_parent.z) +
          (cidx_child.y - 2 * cidx_parent.y) * 2 +
          (cidx_child.x - 2 * cidx_parent.x) * 4 ;
        dtype *de2dc_cur = &de2dc_[de2dc_index * de2dc_r_padded * de2dc_c];

        /* dgemv */
        for(j = child_tid; j < de2dc_r_padded; j += num_thr_per_child) {
          dtype tmp = trg_dwn_chk_val_[i * sp_dc_n_padded + j];
          for(k = 0; k < de2dc_c; k++) {
            tmp += de2dc_cur[k * de2dc_r_padded + j] * 
              tmpDen_[child_id * sp_de_n_padded + k];
          }
          tmpDen[child_id * de2dc_r_padded + j] = tmp;
        }


        /* dgemv */
        for(j = child_tid; j < dc2de_r_padded; j += num_thr_per_child) {
          dtype tmp = 0.0;
          for(k = 0; k < dc2de_c; k++) {
            tmp += dc2de_[k * dc2de_r_padded + j] * 	
              tmpDen[child_id * de2dc_r_padded + k];
          }
          tmpDen__[child_id * dc2de_r_padded + j] = tmp;
        }


        /* scale */
        sclvec = __powf (2.0, -node_depth);
        for(j = child_tid; j < sp_de_n_padded; j += num_thr_per_child) {
          trg_dwn_equ_den_[i * sp_de_n_padded + j] = 
            tmpDen__[child_id * sp_de_n_padded + j] * sclvec;
          // if(i==585) printf("%d %f\n", j, trg_dwn_equ_den_[i * sp_de_n_padded + j]);
        }
      }
    }
  }

}


__global__
  void
down_eval__gpu_(int num_thr_per_child, int n_boxes_, int offset, int* children,
    int3 *path2Node, int max_depth,
    dtype *trg_dwn_chk_val_, int sp_dc_n, int sp_dc_n_padded,
    dtype *trg_dwn_equ_den_, int sp_de_n, int sp_de_n_padded,
    dtype *dc2de_, int dc2de_r, int dc2de_r_padded, int dc2de_c,
    dtype *de2dc_, int de2dc_r, int de2dc_r_padded, int de2dc_c)
{
  /* at minimum size of 8x SP[DE].n padded */
  __shared__ dtype tmpDen[SP_DE_N_PADDED_8];
  /* at minimum size of 8x SP[DE].n padded */
  __shared__ dtype tmpDen_[SP_DE_N_PADDED_8];
  __shared__ dtype tmpDen__[SP_DE_N_PADDED_8];

  int i, j, k;	
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;



  if(bid < n_boxes_) {
#if 0
    for(i = tid; i < SP_DE_N_PADDED_8; i += blockDim.x) {
      tmpDen[i] = 0.0;
      tmpDen_[i] = 0.0;
      tmpDen__[i] = 0.0;
    }
    __syncthreads ();
#endif

    bid += offset;
    int node_depth = 4;

    dtype sclvec = __powf (2.0, node_depth);
    for(i = tid; i < sp_de_n_padded; i += blockDim.x) {
      tmpDen[i] = trg_dwn_equ_den_[bid * sp_de_n_padded + i] * sclvec;
      //if(bid==4680) printf("||| %d %f\n", i, tmpDen[i]);
      // if(bid==617) printf("||| %d %f\n", i, trg_dwn_equ_den_[bid * sp_de_n_padded + i]);
    }
    __syncthreads ();


    /* 0 ~ 7 */
    int child_id = tid / num_thr_per_child;
    int child_tid = tid % num_thr_per_child;
    int cur_child_id = children[bid] + child_id;

    int3 cidx_parent = path2Node[bid];
    int3 cidx_child = path2Node[cur_child_id];
    int de2dc_index = (cidx_child.z - 2 * cidx_parent.z) + 
      (cidx_child.y - 2 * cidx_parent.y) * 2 +
      (cidx_child.x - 2 * cidx_parent.x) * 4;
    dtype *de2dc_cur = &de2dc_[de2dc_index * de2dc_r_padded * de2dc_c];
    /* dgemv */
    for(i = child_tid; i < de2dc_r_padded; i += num_thr_per_child) {
      dtype tmp = trg_dwn_chk_val_[cur_child_id * sp_dc_n_padded + i];
      // dtype tmp1 = 0.0;
      for(j = 0; j < de2dc_c; j++) {
        tmp += de2dc_cur[j * de2dc_r_padded + i] * tmpDen[j];
        // tmp1 += de2dc_cur[j * de2dc_r_padded + i] * tmpDen[j];
        /*
           if(cur_child_id==37447 && i==0) 
           printf("%d %f x %f += %f\n", j, de2dc_cur[j * de2dc_r_padded + i],
           tmpDen[j], tmp1);
         */
      }
      // tmpDen_[child_id * de2dc_r_padded + i] = tmp + tmp1;
      tmpDen_[child_id * de2dc_r_padded + i] = tmp;
      // if(cur_child_id==37447) printf("%d %d %d %f %f\n", bid, i, de2dc_index, tmp1, tmp);
    }
    __syncthreads ();

    /* dgemv */
    for(i = child_tid; i < dc2de_r_padded; i += num_thr_per_child) {
      dtype tmp = 0.0;
      for(j = 0; j < dc2de_c; j++) {
        tmp += dc2de_[j * dc2de_r_padded + i] * 
          tmpDen_[child_id * de2dc_r_padded + j];
      }
      tmpDen__[child_id * dc2de_r_padded + i] = tmp;
      // if(cur_child_id==4681) printf("%d %d %f\n", bid, i, tmp);
    }
    __syncthreads ();

    /* scale */
    node_depth++;
    sclvec = __powf (2.0, -node_depth);
    for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
      tmpDen_[child_id * sp_de_n_padded + i] =
        tmpDen__[child_id * sp_de_n_padded + i] * sclvec;	
      // if(cur_child_id==4681) printf("%d %d %f\n", bid, i, tmpDen_[child_id * sp_de_n_padded + i]);
    }
    __syncthreads ();

    /* level 5 is last */
    if(node_depth == max_depth) {
      for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
        trg_dwn_equ_den_[cur_child_id * sp_de_n_padded + i] = 
          tmpDen_[child_id * sp_de_n_padded + i];
        // if(cur_child_id==4681) printf("%d %d %f\n", bid, child_tid, trg_dwn_equ_den_[cur_child_id * sp_de_n_padded + child_tid]);
      }
    } else {
      /* continue down to next level (depth 6) */
      int first_child = children[cur_child_id];

      /* scale */
      sclvec = __powf (2.0, node_depth);
      for(i = child_tid; i < sp_de_n_padded; i += num_thr_per_child) {
        tmpDen_[child_id * sp_de_n_padded + i] = 
          tmpDen_[child_id * sp_de_n_padded + i] * sclvec;
      }
      node_depth++;

      /* go through each child */
      for(i = first_child ; i < first_child + 8; i++) {
        cidx_parent = path2Node[cur_child_id];
        cidx_child = path2Node[i];
        de2dc_index = (cidx_child.z - 2 * cidx_parent.z) +
          (cidx_child.y - 2 * cidx_parent.y) * 2 +
          (cidx_child.x - 2 * cidx_parent.x) * 4 ;
        dtype *de2dc_cur = &de2dc_[de2dc_index * de2dc_r_padded * de2dc_c];

        /* dgemv */
        for(j = child_tid; j < de2dc_r_padded; j += num_thr_per_child) {
          dtype tmp = trg_dwn_chk_val_[i * sp_dc_n_padded + j];
          for(k = 0; k < de2dc_c; k++) {
            tmp += de2dc_cur[k * de2dc_r_padded + j] * 
              tmpDen_[child_id * sp_de_n_padded + k];
          }
          tmpDen[child_id * de2dc_r_padded + j] = tmp;
        }


        /* dgemv */
        for(j = child_tid; j < dc2de_r_padded; j += num_thr_per_child) {
          dtype tmp = 0.0;
          for(k = 0; k < dc2de_c; k++) {
            tmp += dc2de_[k * dc2de_r_padded + j] * 	
              tmpDen[child_id * de2dc_r_padded + k];
          }
          tmpDen__[child_id * dc2de_r_padded + j] = tmp;
        }


        /* scale */
        sclvec = __powf (2.0, -node_depth);
        for(j = child_tid; j < sp_de_n_padded; j += num_thr_per_child) {
          trg_dwn_equ_den_[i * sp_de_n_padded + j] = 
            tmpDen__[child_id * sp_de_n_padded + j] * sclvec;
        }
      }
    }
  }
}

#if 0
/* should be equal to SP[DE].n padded*/
/* NP=3 ==> 32 */
/* NP=3 ==> 64 */
//#define SL_POS_SIZE 32
#define SL_POS_SIZE 64
#endif

__global__
  void
down_eval__gpu_leaf (int n_boxes_, int *Bptr_, int *Bn_,
    dtype *x_, dtype *y_, dtype *z_, dtype *w_,
    dtype *sp_de_, int sp_de_n, int sp_de_n_padded,
    dtype *trg_dwn_equ_den_,
    dtype *radius_, dtype *center0_, dtype *center1_,
    dtype *center2_, int offset)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

  if(bid < n_boxes_) {
    __shared__ dtype SL_POS[4][SL_POS_SIZE];

    int start;
    int end;

    /* first and last point in this box */
    start = Bptr_[bid];
    end = start + Bn_[bid];

    /* if there are any points in the box (leaf node) */
    if(start < end) {
      int i, j;
      dtype c0, c1, c2, r;
      /* find center and rdaius */
      c0 = center0_[bid - offset];
      c1 = center1_[bid - offset];
      c2 = center2_[bid - offset];
      r = radius_[bid - offset];
      /* for each thread, do 1 point in sl_pos */
      for(i = tid; i < sp_de_n_padded; i += blockDim.x) {
        SL_POS[0][i] = c0 + r * sp_de_[i];
        SL_POS[1][i] = c1 + r * sp_de_[sp_de_n_padded + i];
        SL_POS[2][i] = c2 + r * sp_de_[2 * sp_de_n_padded + i];
        SL_POS[3][i] = trg_dwn_equ_den_[bid * sp_de_n_padded + i];
      }
      __syncthreads ();

      /* ulist calc */
      /* for each point in target */
      for(i = tid; i < Bn_[bid]; i += blockDim.x) {
        dtype tx = x_[start + i];
        dtype ty = y_[start + i];
        dtype tz = z_[start + i];
        dtype td = 0.0;
        /* for each point in sl_pos */
        for(j = 0; j < sp_de_n; j++) {
          dtype x = tx - SL_POS[0][j];
          dtype y = ty - SL_POS[1][j];
          dtype z = tz - SL_POS[2][j];
          dtype rsq = (x * x) + (y * y) + (z * z);
          rsq = rsqrt (rsq);
          td += SL_POS[3][j] * rsq;
        }
        w_[start + i] += OOFP_R * td;
        // if(bid == 73) printf("%d %f\n", tid, OOFP_R * td);
      }
    }
  }
}

/* ------------------------------------------------------------------------
 */
__global__
void
down_eval__gpu_depth_2 (int n_boxes_,
												int *tag_,
												int sp_dc_n_padded,
												dtype *trg_dwn_chk_val_,
												int dc2de_c, 
												int dc2de_r,
												int dc2de_r_padded,
												dtype *dc2de_,
												int sp_de_n_padded,
												dtype *trg_dwn_equ_den_
											 )
{
	int i, j; 
	int node_depth = 2; 
	int tid = threadIdx.x;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;

  __shared__ dtype tmpDen[SP_DE_N_PADDED_8];

	if(bid < n_boxes_) {
		if(tag_[bid] & LET_TRGNODE) {
			/* dgemv */
			for(i = tid; i < dc2de_r_padded; i += blockDim.x) {
				dtype tmp = 0.0;
				for(j = 0; j < dc2de_c; j++) {
					tmp += dc2de_[j * dc2de_r_padded + i] * 
								 trg_dwn_chk_val_[bid * sp_dc_n_padded + j];
				}
				tmpDen[i] = tmp;
			}
			
			/* scale */
			dtype sclvec = __powf (2.0, -node_depth);
			for(i = tid; i < sp_de_n_padded; i+= blockDim.x) {
				trg_dwn_equ_den_[bid * sp_de_n_padded + i] = tmpDen[i] * sclvec;
			}
		}
	}
}

__global__
void
down_eval__gpu_non_leaves (int n_boxes_,
													 int node_depth,
												 	 int *tag_,
													 int *depth_,
													 int3 *path2Node_,
													 int *parent_,
													 int sp_dc_n_padded,
													 dtype *trg_dwn_chk_val_,
													 int dc2de_c, 
													 int dc2de_r,
													 int dc2de_r_padded,
													 dtype *dc2de_,
													 int de2dc_c,
													 int de2dc_r,
													 int de2dc_r_padded,
													 dtype *de2dc_,
													 int sp_de_n_padded,
													 dtype *trg_dwn_equ_den_
											 )

{
	int i, j; 
	int tid = threadIdx.x;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;

	__shared__ dtype tmpDen[SP_DE_N_PADDED_8];
	__shared__ dtype tmpDen_[SP_DE_N_PADDED_8];

	if(bid < n_boxes_) {
		if(tag_[bid] & LET_TRGNODE && depth_[bid] == node_depth) {
			int parent_bid = parent_[bid];

			/* scale */
			dtype sclvec = __powf (2.0, (node_depth - 1));
			for(i = tid; i < sp_de_n_padded; i += blockDim.x) {
				tmpDen[i] = trg_dwn_equ_den_[parent_bid * sp_de_n_padded + i] * sclvec;
			}	
			__syncthreads ();


			/* dgemv */
			int3 cidx_parent = path2Node_[parent_bid];
			int3 cidx_child = path2Node_[bid];
			int de2dc_index = (cidx_child.z - 2 * cidx_parent.z) + 
												(cidx_child.y - 2 * cidx_parent.y) * 2 + 
												(cidx_child.x - 2 * cidx_parent.x) * 4;
			dtype *de2dc_cur = &de2dc_[de2dc_index * de2dc_r_padded * de2dc_c]; 
			for(i = tid; i < de2dc_r_padded; i += blockDim.x) {
				dtype tmp = trg_dwn_chk_val_[bid * sp_dc_n_padded + i];
				for(j = 0; j < de2dc_c; j++) {
					tmp += de2dc_cur[j * de2dc_r_padded + i] * tmpDen[j];
				}
				tmpDen_[i] = tmp;
			}
			__syncthreads ();


			/* dgemv */
			for(i = tid; i < dc2de_r; i += blockDim.x) {
				dtype tmp = 0.0;
				for(j = 0; j < dc2de_c; j++) {
					tmp += dc2de_[j * dc2de_r_padded + i] * tmpDen_[j];
				}
				tmpDen[i] = tmp;
			}
			__syncthreads ();


			/* scale and write back */
			sclvec = __powf (2.0, -node_depth);	
			for(i = tid; i < sp_de_n_padded; i += blockDim.x) {
				trg_dwn_equ_den_[bid * sp_de_n_padded + i] = tmpDen[i] * sclvec;	
			}
		}
	}
}


__global__
void
down_eval__gpu_leaves (int n_boxes_,
											 int *Bptr_,
											 int *Bn_,
											 dtype *x_,
											 dtype *y_,
											 dtype *z_,
											 dtype *w_,
											 int sp_de_n,
											 int sp_de_n_padded,
											 dtype *sp_de_,
											 dtype *trg_dwn_equ_den_,
											 dtype *radius_,
											 dtype *center0_,
											 dtype *center1_,
											 dtype *center2_
											 )
{
	int tid = threadIdx.x;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;

	int i, j;
	int start, end;

	__shared__ dtype SL_POS[4][SL_POS_SIZE];

	if(bid < n_boxes_) {
		start = Bptr_[bid];
		end = start + Bn_[bid];

		if(start < end) {
			dtype c0, c1, c2, r;
			/* find center and rdaius */
			c0 = center0_[bid];
			c1 = center1_[bid];
			c2 = center2_[bid];
			r = radius_[bid];
			/* for each thread, do 1 point in sl_pos */
			for(i = tid; i < sp_de_n_padded; i += blockDim.x) {
				SL_POS[0][i] = c0 + r * sp_de_[i];
				SL_POS[1][i] = c1 + r * sp_de_[sp_de_n_padded + i];
				SL_POS[2][i] = c2 + r * sp_de_[2 * sp_de_n_padded + i];
				SL_POS[3][i] = trg_dwn_equ_den_[bid * sp_de_n_padded + i];
			}
			__syncthreads ();

			/* ulist calc */
			/* for each point in target */
			for(i = tid; i < Bn_[bid]; i += blockDim.x) {
				dtype tx = x_[start + i];
				dtype ty = y_[start + i];
				dtype tz = z_[start + i];
				dtype td = 0.0;
				/* for each point in sl_pos */
				for(j = 0; j < sp_de_n; j++) {
					dtype x = tx - SL_POS[0][j];
					dtype y = ty - SL_POS[1][j];
					dtype z = tz - SL_POS[2][j];
					dtype rsq = (x * x) + (y * y) + (z * z);
					rsq = rsqrt (rsq);
					td += SL_POS[3][j] * rsq;
				}
				w_[start + i] += OOFP_R * td;
      }
		}		
	}

}

/* ------------------------------------------------------------------------
 */

int
d2d__gpu (FMMWrapper_t *f)
{
	int i;

	AllNodes *All_N = f->AN;
	vector<NodeTree>& nodeVec = *All_N->N;

  /* do level 0~4 
   * nothing is done for level 0 and 1 */
  const int NB = get_thread_block_size_wlist ();
  const int NG = nodeVec.size ();

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

	/* do non-leaves */
	/* first do level 2 */
	down_eval__gpu_depth_2 <<<GB, TB>>> (nodeVec.size (),
																			 f->tag_d_,
																			 f->SP_DC_n_padded_,
																			 f->TRG_DWN_CHK_VAL_d_,
																			 f->DC2DE_c, 
																			 f->DC2DE_r,
																			 f->DC2DE_r_padded,
																			 f->DC2DE_d_,
																			 f->SP_DE_n_padded,
																			 f->TRG_DWN_EQU_DEN_d_);
	hipDeviceSynchronize ();
	gpu_check_error (stderr);


	/* do level 3 ~ last */
	for(i = 3; i <= f->tree_max_depth; i++) {
		/* 	
			input: tval = trg_dwn_chk_val
			output: dden = trg_dwn_equ_den
			algorithm: 	
				tmpDen = trg_dwn_equ_den[parent] * sclvec
				trg_dwn_chk_val[i] += dgemv (DE2DC, tmpDen)
				tmpDen = dgemv (DC2DE, tval[i])
				trg_dwn_equ_den[i] = tmpDen * sclvec
		 */	
		down_eval__gpu_non_leaves <<<GB, TB>>> (nodeVec.size (),
																						i,
																						f->tag_d_,
																						f->depth_d_,
																						f->path2Node_d_,
																						f->parent_d_,
																						f->SP_DC_n_padded_,
																						f->TRG_DWN_CHK_VAL_d_,
																						f->DC2DE_c,
																						f->DC2DE_r,
																						f->DC2DE_r_padded,
																						f->DC2DE_d_,
																						f->DE2DC_c,
																						f->DE2DC_r,
																						f->DE2DC_r_padded,
																						f->DE2DC_d_,
																						f->SP_DE_n_padded,
																						f->TRG_DWN_EQU_DEN_d_);
	  hipDeviceSynchronize ();
	  gpu_check_error (stderr);

	}

	return 0;
}

int
d2t__gpu (FMMWrapper_t *f)
{
	AllNodes *All_N = f->AN;
	vector<NodeTree>& nodeVec = *All_N->N;

  const int NB = get_thread_block_size_wlist ();
  const int NG = nodeVec.size ();

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

	/* do leaves */
	down_eval__gpu_leaves <<<GB, TB>>> (nodeVec.size (),
																			f->T_d_.Bptr_,
																			f->T_d_.Bn_,
																			f->T_d_.x_,
																			f->T_d_.y_,
																			f->T_d_.z_,
																			f->T_d_.w_,
																			f->SP_DE_n_,
																			f->SP_DE_n_padded,
																			f->SP_DE_d_,
																			f->TRG_DWN_EQU_DEN_d_,
																			f->radius_d_,
																			f->center0_d_,
																			f->center1_d_,
																			f->center2_d_
																			);
	hipDeviceSynchronize ();
	gpu_check_error (stderr);

	return 0;
}

int
down_calc__gpu (FMMWrapper_t *f)
{
	int i;

	AllNodes *All_N = f->AN;
	vector<NodeTree>& nodeVec = *All_N->N;

  /* do level 0~4 
   * nothing is done for level 0 and 1 */
  const int NB = get_thread_block_size_wlist ();
  const int NG = nodeVec.size ();

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

	/* do non-leaves */
	/* first do level 2 */
	down_eval__gpu_depth_2 <<<GB, TB>>> (nodeVec.size (),
																			 f->tag_d_,
																			 f->SP_DC_n_padded_,
																			 f->TRG_DWN_CHK_VAL_d_,
																			 f->DC2DE_c, 
																			 f->DC2DE_r,
																			 f->DC2DE_r_padded,
																			 f->DC2DE_d_,
																			 f->SP_DE_n_padded,
																			 f->TRG_DWN_EQU_DEN_d_);
	hipDeviceSynchronize ();
	gpu_check_error (stderr);

	/* do level 3 ~ last */
	for(i = 3; i <= f->tree_max_depth; i++) {
		/* 	
			input: tval = trg_dwn_chk_val
			output: dden = trg_dwn_equ_den
			algorithm: 	
				tmpDen = trg_dwn_equ_den[parent] * sclvec
				trg_dwn_chk_val[i] += dgemv (DE2DC, tmpDen)
				tmpDen = dgemv (DC2DE, tval[i])
				trg_dwn_equ_den[i] = tmpDen * sclvec
		 */	

		down_eval__gpu_non_leaves <<<GB, TB>>> (nodeVec.size (),
																						i,
																						f->tag_d_,
																						f->depth_d_,
																						f->path2Node_d_,
																						f->parent_d_,
																						f->SP_DC_n_padded_,
																						f->TRG_DWN_CHK_VAL_d_,
																						f->DC2DE_c,
																						f->DC2DE_r,
																						f->DC2DE_r_padded,
																						f->DC2DE_d_,
																						f->DE2DC_c,
																						f->DE2DC_r,
																						f->DE2DC_r_padded,
																						f->DE2DC_d_,
																						f->SP_DE_n_padded,
																						f->TRG_DWN_EQU_DEN_d_);
	  hipDeviceSynchronize ();
	  gpu_check_error (stderr);

	}

	/* do leaves */
	down_eval__gpu_leaves <<<GB, TB>>> (nodeVec.size (),
																			f->T_d_.Bptr_,
																			f->T_d_.Bn_,
																			f->T_d_.x_,
																			f->T_d_.y_,
																			f->T_d_.z_,
																			f->T_d_.w_,
																			f->SP_DE_n_,
																			f->SP_DE_n_padded,
																			f->SP_DE_d_,
																			f->TRG_DWN_EQU_DEN_d_,
																			f->radius_d_,
																			f->center0_d_,
																			f->center1_d_,
																			f->center2_d_
																			);
	hipDeviceSynchronize ();
	gpu_check_error (stderr);

  return 0;
}

__global__
void
wlist_eval__gpu (int n_boxes_,
								 int *tag_,
								 int *srcNum_,
								 int *child_,
								 int *Bptr_T_,
								 int *Bn_T_,
								 dtype *x_T_,
								 dtype *y_T_,
								 dtype *z_T_,
								 dtype *w_T_,
								 int *Bptr_S_,
								 int *Bn_S_,
								 dtype *x_S_,
								 dtype *y_S_,
								 dtype *z_S_,
								 dtype *w_S_,
								 int *L__w,
								 int *Ptr__w,
								 int sp_ue_n_,
								 int sp_ue_n_padded,
								 dtype *sp_ue_,
								 dtype *radius_,
								 dtype *center0_,
								 dtype *center1_,
								 dtype *center2_,
								 dtype *src_upw_equ_den_
								)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

	int wn, k, t, s;

  __shared__ dtype SL_POS[4][SL_POS_SIZE];

	/* valid box */
	if(bid < n_boxes_) {
		/* if target box */
		if(tag_[bid] & LET_TRGNODE) {
			/* if target box is a leaf */
			if(child_[bid] == -1) {
				/* what are this box's target points */
        int trg_begin = Bptr_T_[bid];
        int trg_end = Bptr_T_[bid] + Bn_T_[bid];

				/* what are this box's W list */
        int w_start = Ptr__w[bid];
        int w_end = Ptr__w[bid + 1];

				/* do this only if box has points AND a list of W nodes */
				if(trg_begin < trg_end && w_start < w_end) {
	
					/* for each w nodes in this box's list */
					for(wn = w_start; wn < w_end; wn++) {
						/* node id */
						int src = L__w[wn];

						/* if conditions are met, compute localpos */
						if(child_[src] != -1 || srcNum_[src] >= sp_ue_n_) {
							dtype c0 = center0_[src];
							dtype c1 = center1_[src];
							dtype c2 = center2_[src];
							dtype r = radius_[src];
							for(k = tid; k < sp_ue_n_padded; k += blockDim.x) {
								SL_POS[0][k] = c0 + r * sp_ue_[k];
								SL_POS[1][k] = c1 + r * sp_ue_[sp_ue_n_padded + k];
								SL_POS[2][k] = c2 + r * sp_ue_[2 * sp_ue_n_padded + k];
								SL_POS[3][k] = src_upw_equ_den_[src * sp_ue_n_padded + k];
							}
						}
						__syncthreads ();

						if(child_[src] == -1 && srcNum_[src] < sp_ue_n_) {
							/* go through each target point */
							for(t = trg_begin + tid; t < trg_end; t += blockDim.x) {
								dtype xt = x_T_[t];
								dtype yt = y_T_[t];
								dtype zt = z_T_[t];
								dtype wt = 0.0;
				
								/* go through each source point */
								int src_begin = Bptr_S_[src];
								int src_end = Bptr_S_[src] + Bn_S_[src];
								for(s = src_begin; s < src_end; s++) {
									dtype xs = xt - x_S_[s];
									dtype ys = yt - y_S_[s];
									dtype zs = zt - z_S_[s];
					
									dtype rsq = xs * xs + ys * ys + zs * zs;
									rsq = rsqrt (rsq);

									wt += w_S_[s] * rsq;
								}

								w_T_[t] += wt * OOFP_R;
							}
						} else {
							/* go through each target point */
							for(t = trg_begin + tid; t < trg_end; t += blockDim.x) {
								dtype xt = x_T_[t];
								dtype yt = y_T_[t];
								dtype zt = z_T_[t];
								dtype wt = 0.0;

								/* go through each sl_pos point */
								for(s = 0; s < sp_ue_n_; s++) {
									dtype xs = xt - SL_POS[0][s];
									dtype ys = yt - SL_POS[1][s];
									dtype zs = zt - SL_POS[2][s];

									dtype rsq = xs * xs + ys * ys + zs * zs;
									rsq = rsqrt (rsq);

									wt += SL_POS[3][s] * rsq;
								}
						
								w_T_[t] += wt * OOFP_R;
							}
						}
						__syncthreads ();
					}
				}
			}
		}
	}
}


__global__
void
wlist_eval__gpu_ (int n_boxes_,
								 int *tag_,
								 int *srcNum_,
								 int *child_,
								 int *Bptr_T_,
								 int *Bn_T_,
								 dtype *x_T_,
								 dtype *y_T_,
								 dtype *z_T_,
								 dtype *w_T_,
								 int *Bptr_S_,
								 int *Bn_S_,
								 dtype *x_S_,
								 dtype *y_S_,
								 dtype *z_S_,
								 dtype *w_S_,
								 int *L__w,
								 int *Ptr__w,
								 int sp_ue_n_,
								 int sp_ue_n_padded,
								 dtype *sp_ue_,
								 dtype *radius_,
								 dtype *center0_,
								 dtype *center1_,
								 dtype *center2_,
								 int uc2ue_r,
								 int uc2ue_r_padded,
								 dtype *src_upw_equ_den_
								)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * gridDim.x + blockIdx.x;

  int i, j, k;

  __shared__ dtype SL_POS[4][SL_POS_SIZE];

	/* valid box */
	if(bid < n_boxes_) {
		/* if target box */
		if(tag_[bid] & LET_TRGNODE) {
			/* if target box is a leaf */
			if(child_[bid] == -1) {

				/* loop boundaries */
				int w_start = Ptr__w[bid];
				int w_end = Ptr__w[bid + 1];
				int trg_begin = Bptr_T_[bid];
				int trg_end = Bptr_T_[bid] + Bn_T_[bid];

				/* start only if all loop boundaries have actual work */
				if(w_start < w_end && trg_begin < trg_end) {

					/* each thread gets 1 source */
					for(i = trg_begin + tid; i < trg_end; i += blockDim.x) {
						dtype xt = x_T_[i];
						dtype yt = y_T_[i];
						dtype zt = z_T_[i];

						/* for each target point loop over wlist */
						for(j = w_start; j < w_end; j++) {
							dtype wt = 0.0;
							int src = L__w[j];

							if(child_[src] != -1 || srcNum_[src] >= sp_ue_n_) {
								dtype c0 = center0_[src];
								dtype c1 = center1_[src];
								dtype c2 = center2_[src];
								dtype r = radius_[src];
								for(k = tid; k < sp_ue_n_padded; k += blockDim.x) {
									SL_POS[0][k] = c0 + r * sp_ue_[k];
									SL_POS[1][k] = c1 + r * sp_ue_[sp_ue_n_padded + k];
									SL_POS[2][k] = c2 + r * sp_ue_[2 * sp_ue_n_padded + k];
									SL_POS[3][k] = src_upw_equ_den_[bid * sp_ue_n_padded + k];
								}	
								__syncthreads ();
							}


							if(child_[src] == -1 && srcNum_[src] < sp_ue_n_) {
								// if(bid == 31 && tid == 0) printf("==> %d\n", src);
								int src_begin = Bptr_S_[src];
								int src_end = Bptr_S_[src] + Bn_S_[src];
								/* loop over points in the source box */
								for(k = src_begin; k < src_end; k++) {
									dtype xs = x_S_[k];
									dtype ys = y_S_[k];
									dtype zs = z_S_[k];
									// dtype ws = w_S_[k];

									xs = xt - xs;
									ys = yt - ys;
									zs = zt - zs;

									dtype rsq = xs * xs + ys * ys + zs * zs;
									rsq = rsqrt (rsq);
			
									wt += wt * rsq;
								}
								// wt = wt * OOFP_R;
								w_T_[i] += wt * OOFP_R;
							} else {
							
								/* ulist_calc */
								for(k = 0; k < sp_ue_n_; k++) {
									dtype x = xt - SL_POS[0][k];
									dtype y = yt - SL_POS[1][k];
									dtype z = zt - SL_POS[2][k];
								
									dtype rsq = x * x + y * y + z * z;
									rsq = rsqrt (rsq);
					
									wt += SL_POS[3][k] * rsq;
								}
								// wt = wt * OOFP_R;
								w_T_[i] += wt * OOFP_R;
							} /* if src == -1 && srcNum < SP[UE].n */
						} /* for each wlist neighbor */

						/* write result back */
						// w_T_[i] += wt;
					} /* for each target point */
				} /* if there is work to do */
			}
		}
	}

}

int
wlist_calc__gpu(FMMWrapper_t *f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* do level 0~4 
   * nothing is done for level 0 and 1 */
  const int NB = get_thread_block_size_down ();
  const int NG = nodeVec.size ();

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

	wlist_eval__gpu <<<GB, TB>>> (nodeVec.size (),
																f->tag_d_,
																f->srcNum_d_,
																f->child_d_,
																f->T_d_.Bptr_,
																f->T_d_.Bn_,
																f->T_d_.x_,
																f->T_d_.y_,
																f->T_d_.z_,
																f->T_d_.w_,
																f->S_d_.Bptr_,
																f->S_d_.Bn_,
																f->S_d_.x_,
																f->S_d_.y_,
																f->S_d_.z_,
																f->S_d_.w_,
																f->W_d_.L_,
																f->W_d_.Ptr_,
																f->SP_UE_n_,
																f->SP_UE_n_padded,
																f->SP_UE_d_,
                               	f->radius_d_,
                               	f->center0_d_,
                               	f->center1_d_,
                               	f->center2_d_,
																f->SRC_UPW_EQU_DEN_d_
																);
	hipDeviceSynchronize ();
	gpu_check_error (stderr);


	return 0;
}


__global__
void
xlist_eval__gpu (int n_boxes_,
								 int *tag_,
								 int *trgNum_,
								 int *child_,
								 int *Bptr_T_,
								 int *Bn_T_,
								 dtype *x_T_,
								 dtype *y_T_,
								 dtype *z_T_,
								 dtype *w_T_,
								 int *Bptr_S_,
								 int *Bn_S_,
								 dtype *x_S_,
								 dtype *y_S_,
								 dtype *z_S_,
								 dtype *w_S_,
								 int *L__x,
								 int *Ptr__x,
								 int sp_dc_n_,
								 int sp_dc_n_padded,
								 dtype *sp_dc_,
								 dtype *radius_,
								 dtype *center0_,
								 dtype *center1_,
								 dtype *center2_,
								 dtype *trg_dwn_chk_val_
								)
{
	int tid = threadIdx.x;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;

	int xn, k, t, s;

	__shared__ dtype SL_POS[4][SL_POS_SIZE];

	if(bid < n_boxes_) {
		if(tag_[bid] & LET_TRGNODE) {
			int x_start = Ptr__x[bid];
			int x_end = Ptr__x[bid + 1];
			/* do work if there is a x list */
			if(x_start < x_end) {

				/* for each node in x list */
				for(xn = x_start; xn < x_end; xn++) {
					int src = L__x[xn];

					if(child_[bid] != -1 || trgNum_[bid] >= sp_dc_n_) {
						dtype c0 = center0_[bid];
						dtype c1 = center1_[bid];
						dtype c2 = center2_[bid];
						dtype r = radius_[bid];
						for(k = tid; k < sp_dc_n_padded; k += blockDim.x) {
							SL_POS[0][k] = c0 + r * sp_dc_[k];
							SL_POS[1][k] = c1 + r * sp_dc_[sp_dc_n_padded + k];
							SL_POS[2][k] = c2 + r * sp_dc_[2 * sp_dc_n_padded + k];
							SL_POS[3][k] = trg_dwn_chk_val_[bid * sp_dc_n_padded + k];
						}
					}
					__syncthreads ();

					if(child_[bid] == -1 && trgNum_[bid] < sp_dc_n_) {
						int trg_begin = Bptr_T_[bid];
						int trg_end = Bptr_T_[bid] + Bn_T_[bid];
						for(t = trg_begin + tid; t < trg_end; t += blockDim.x) {
							dtype xt = x_T_[t];
							dtype yt = y_T_[t];
							dtype zt = z_T_[t];
							dtype wt = 0.0;

							int src_begin = Bptr_S_[src];
							int src_end = Bptr_S_[src] + Bn_S_[src];
							for(s = src_begin; s < src_end; s++) {
								dtype xs = xt - x_S_[s];
								dtype ys = yt - y_S_[s];
								dtype zs = zt - z_S_[s];

								dtype rsq = xs * xs + ys * ys + zs * zs;
								rsq = rsqrt (rsq);

								wt += w_S_[s] * rsq;
							}
							w_T_[t] += wt * OOFP_R;
						}
					} else {
						for(t = tid; t < sp_dc_n_; t += blockDim.x) {
							dtype xt = SL_POS[0][t];
							dtype yt = SL_POS[1][t];
							dtype zt = SL_POS[2][t];
							dtype wt = 0.0;

							int src_begin = Bptr_S_[src];
							int src_end = Bptr_S_[src] + Bn_S_[src];
							for(s = src_begin;  s < src_end; s++) {
								dtype xs = xt - x_S_[s];
								dtype ys = yt - y_S_[s];
								dtype zs = zt - z_S_[s];
			
								dtype rsq = xs * xs + ys * ys + zs * zs;
								rsq = rsqrt (rsq);

								wt += w_S_[s] * rsq;
							}
							trg_dwn_chk_val_[bid * sp_dc_n_padded + t] = SL_POS[3][t] + 
																													 wt * OOFP_R;
						}
					}
					__syncthreads ();
				}
			} /* if xlist is not 0 */
		} /* if box is target node */
	} /* if bid is withint nodeVec.size () */
}

int
xlist_calc__gpu(FMMWrapper_t *f)
{

  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* do level 0~4 
   * nothing is done for level 0 and 1 */
  const int NB = get_thread_block_size_down ();
  const int NG = nodeVec.size ();

  dim3 GB (65535, (NG / 65535) + 1, 1);
  dim3 TB (NB, 1, 1);

	xlist_eval__gpu <<<GB, TB>>> (nodeVec.size (),
																f->tag_d_,
																f->trgNum_d_,
																f->child_d_,
																f->T_d_.Bptr_,
																f->T_d_.Bn_,
																f->T_d_.x_,
																f->T_d_.y_,
																f->T_d_.z_,
																f->T_d_.w_,
																f->S_d_.Bptr_,
																f->S_d_.Bn_,
																f->S_d_.x_,
																f->S_d_.y_,
																f->S_d_.z_,
																f->S_d_.w_,
																f->X_d_.L_,
																f->X_d_.Ptr_,
																f->SP_DC_n_,
																f->SP_DC_n_padded_,
																f->SP_DC_d_,
                               	f->radius_d_,
                               	f->center0_d_,
                               	f->center1_d_,
                               	f->center2_d_,
																f->TRG_DWN_CHK_VAL_d_
																);
	hipDeviceSynchronize ();
	gpu_check_error (stderr);

	return 0;
}

int
copy_trg_val__gpu (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;
  /* transfer data back */
  hipMemcpy (f->T_h_.w_, f->T_d_.w_, f->T_h_.Bptr_[f->T_h_.n_boxes_] * 
      sizeof (dtype), hipMemcpyDeviceToHost);

  /* convert gpu results to cpu */
  for(int i = 0; i < f->T_h_.n_boxes_; i++) {
    for(int j = f->T_h_.Bptr_[i]; j < f->T_h_.Bptr_[i + 1]; j++) {
      All_N->Nt[i].den_pot[j - f->T_h_.Bptr_[i]] += f->T_h_.w_[j];
    }
  }
  for (int i = 0; i < nodeVec.size(); i++) 
	  if( nodeVec[i].tag & LET_TRGNODE)  
      if (nodeVec[i].child == -1) {
        set_value (nodeVec[i].trgNum, All_N->pot_orig, All_N->Nt[i].den_pot, nodeVec[i].trgOwnVecIdxs);
		  }

  return 0;
}

/* eof */
