
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#include "evaluate.h"
#include "util.h"
#include "reals.h"
#include "partial.h"
#include "../timing/timing.h"
#include "node_gpu.h"
#include <cutil_inline.h>
#include <omp.h>

int
run (FMMWrapper_t *F)
{
  struct stopwatch_t* timer = NULL;
  long double t_total;
  timer = stopwatch_create ();

  stopwatch_start (timer);
  fprintf (stderr, "Performing Ulist on the CPU\n");
  omp_set_nested (1);
#pragma omp parallel shared(F) num_threads(2)
  {
#pragma omp sections
    {
      /* GPU section */
#pragma omp section
      {
        up_calc__gpu (F);
        vlist_calc__gpu (F);
				// xlist_calc__gpu (F);
				// wlist_calc__gpu (F);
        down_calc__gpu (F);
        hipDeviceSynchronize ();
      }

      /* CPU section */
#pragma omp section
      {
        ulist_calc__cpu (F);
      }
    }
  }
  copy_trg_val__gpu (F);
  t_total = stopwatch_stop (timer);

  fprintf(stderr, "=== Statistics ===\n");
  fprintf (stdout, "Code: %s\n", get_implementation_name ());
  fprintf (stdout, "Floating-point word size: %lu bytes\n", sizeof (real_t));
  fprintf (stderr, "== Timing for FMM3d ==\n");
  fprintf (stderr, "  ==> Total Execution Time: %Lg secs\n", t_total);

  stopwatch_destroy (timer);

  return 0;
}
/* ------------------------------------------------------------------------
 */
