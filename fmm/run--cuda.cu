
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#include "evaluate.h"
#include "util.h"
#include "reals.h"
#include "partial.h"
#include "../timing/timing.h"
#include "node_gpu.h"
#include <cutil_inline.h>
#include <omp.h>

/* ------------------------------------------------------------------------
 */
/*const char *
get_implementation_name (void)
{
  return "cuda";
}
*/
/* ------------------------------------------------------------------------
 */

int
run (FMMWrapper_t *F)
{
  struct stopwatch_t* timer = NULL;
  long double t_up, t_u, t_v, t_w, t_x, t_down, t_total;
	long double t_u_start, t_w_start, t_x_start, t_down_start;

	long double t_v1_start, t_v2_start, t_v3_start, t_v4_start, t_copy_start;
	long double t_v1, t_v2, t_v3, t_v4, t_copy;
  timer = stopwatch_create ();

	#if MIN_DATA
		alloc__SOURCE_BOX__ (F);
		alloc__RADIUS__ (F);
		alloc__CENTER__ (F);
		alloc__SP_UC__ (F);
		alloc__UC2UE__ (F);
		alloc__DEPTH__ (F);
		alloc__SRC_UPW_EQU_DEN__ (F);
		alloc__CHILDREN__ (F);
		alloc__UE2UC__ (F);
		alloc__TAG__ (F);

		xfer__SOURCE_BOX__ (F);
		xfer__RADIUS__ (F);
		xfer__CENTER__ (F);
		xfer__SP_UC__ (F);
		xfer__UC2UE__ (F);
		xfer__DEPTH__ (F);
		xfer__CHILDREN__ (F);
		xfer__UE2UC__ (F);
		xfer__TAG__ (F);
	#endif
  fprintf (stderr, "Performing Up calculation ...\n"); 
  stopwatch_start (timer);
  up_calc__gpu (F);
  t_up = stopwatch_elapsed (timer);
  fprintf (stderr, "Done Up.\n");
	#if MIN_DATA
		/* only SRC_UPW_EQU_DEN requires backing up */		
		xfer__SRC_UPW_EQU_DEN__back (F);

		free__SOURCE_BOX__ (F);
		free__RADIUS__ (F);
		free__CENTER__ (F);
		free__SP_UC__ (F);
		free__UC2UE__ (F);
		free__DEPTH__ (F);
		free__SRC_UPW_EQU_DEN__ (F);
		free__CHILDREN__ (F);
		free__UE2UC__ (F);
		free__TAG__ (F);
	#endif

	#if MIN_DATA
		alloc__SOURCE_BOX__ (F);
		alloc__TARGET_BOX__ (F);
		alloc__U_LIST__ (F);

		xfer__SOURCE_BOX__ (F);
		xfer__TARGET_BOX__ (F);
		xfer__U_LIST__ (F);
	#endif
  fprintf (stderr, "Performing U-list calculation (direct evaluation)...\n");
  t_u_start = stopwatch_elapsed (timer);
  ulist_calc__gpu (F);
	hipDeviceSynchronize ();
  t_u = stopwatch_elapsed (timer);
  fprintf (stderr, "Done Ulist.\n");
	#if MIN_DATA
		/* backup TARGET */
		xfer__TARGET_BOX__back (F);

		free__SOURCE_BOX__ (F);
		free__TARGET_BOX__ (F);
		free__U_LIST__ (F);
	#endif

  fprintf (stderr, "Performing V-list calculation (pointwise multiply)...\n");
  // vlist_calc__gpu (F);

  AllNodes *All_N = F->AN;
	#if MIN_DATA
		alloc__DEPTH__ (F);
		alloc__SRC_UPW_EQU_DEN__ (F);
		alloc__REG_DEN__ (F);
		alloc__VLIST_SRC__ (F);

		xfer__DEPTH__ (F);
		xfer__SRC_UPW_EQU_DEN__ (F);
	#endif
  t_v1_start = stopwatch_elapsed (timer);
  compute_fft_src__gpu (F, All_N);
	hipDeviceSynchronize ();
  t_v1 = stopwatch_elapsed (timer);
	#if MIN_DATA
		// this is freed within compute_ifft_src__gpu
		// free__SRC_UPW_EQU_DEN__ (F);
		free__DEPTH__ (F);
		free__REG_DEN__ (F);	
	#endif

	#if MIN_DATA
		alloc__TT__ (F);
		alloc__VLIST_TRANS__ (F);
	#endif
  t_v2_start = stopwatch_elapsed (timer);
  compute_fft_trans__gpu (F, All_N);
	hipDeviceSynchronize ();
  t_v2 = stopwatch_elapsed (timer);
	#if MIN_DATA
		free__TT__ (F);
	#endif

	#if MIN_DATA
		alloc__VLIST_TRG__ (F);
		alloc__VLIST_TLIST__ (F);

		xfer__VLIST_TLIST__ (F);
	#endif
  t_v3_start = stopwatch_elapsed (timer);
  vlist_calc__gpu_ (F, All_N);
	hipDeviceSynchronize ();
  t_v3 = stopwatch_elapsed (timer);
	#if MIN_DATA
		free__VLIST_SRC__ (F);
		free__VLIST_TRANS__ (F);
		free__VLIST_TLIST__ (F);
	#endif

	#if MIN_DATA
		alloc__REG_DEN__ (F);
		// this is created within compute_ifft_trg__gpu
		// alloc__TRG_DWN_CHK_VAL__ (F);
	#endif
  t_v4_start = stopwatch_elapsed (timer);
  compute_ifft_trg__gpu (F, All_N);
	hipDeviceSynchronize ();
  t_v4 = stopwatch_elapsed (timer);
	#if MIN_DATA
		/* backup TRG_DWN_CHK_VAL */
		xfer__TRG_DWN_CHK_VAL__back (F);

		free__VLIST_TRG__ (F);
		free__REG_DEN__ (F);
		free__TRG_DWN_CHK_VAL__ (F);
	#endif

  // t_v = stopwatch_elapsed (timer);
  fprintf (stderr, "Done Vlist.\n ");

  fprintf (stderr, "Performing W-list calculation...\n");
	#if MIN_DATA
		alloc__TAG__ (F);
		alloc__SRCNUM__ (F);
		alloc__CHILDREN__ (F);
		alloc__TARGET_BOX__ (F);
		alloc__SOURCE_BOX__ (F);
		alloc__W_LIST__ (F);
		alloc__SP_UE__ (F);
		alloc__RADIUS__ (F);
		alloc__CENTER__ (F);
		alloc__SRC_UPW_EQU_DEN__ (F);

		xfer__TAG__ (F);
		xfer__SRCNUM__ (F);
		xfer__CHILDREN__ (F);
		xfer__TARGET_BOX__ (F);
		xfer__SOURCE_BOX__ (F);
		xfer__SP_UE__ (F);
		xfer__W_LIST__ (F);
		xfer__RADIUS__ (F);
		xfer__CENTER__ (F);
		xfer__SRC_UPW_EQU_DEN__ (F);
	#endif
  t_w_start = stopwatch_elapsed (timer);
	wlist_calc__gpu (F);
	hipDeviceSynchronize ();
  t_w = stopwatch_elapsed (timer);
	#if MIN_DATA
		/* backup TARGET */
		xfer__TARGET_BOX__back (F);

		free__TAG__ (F);
		free__SRCNUM__ (F);
		free__CHILDREN__ (F);
		free__TARGET_BOX__ (F);
		free__SOURCE_BOX__ (F);
		free__W_LIST__ (F);
		free__SP_UE__ (F);
		free__RADIUS__ (F);
		free__CENTER__ (F);
		free__SRC_UPW_EQU_DEN__ (F);
	#endif
  fprintf (stderr, "Done Wlist.\n ");
 
  fprintf (stderr, "Performing X-list calculation...\n");
	#if MIN_DATA
		alloc__TAG__ (F);
		alloc__TRGNUM__ (F);
		alloc__CHILDREN__ (F);
		alloc__TARGET_BOX__ (F);
		alloc__SOURCE_BOX__ (F);
		alloc__X_LIST__ (F);
		alloc__SP_DC__ (F);
		alloc__RADIUS__ (F);
		alloc__CENTER__ (F);
		alloc__TRG_DWN_CHK_VAL__ (F);

		xfer__TAG__ (F);
		xfer__TRGNUM__ (F);
		xfer__CHILDREN__ (F);
		xfer__TARGET_BOX__ (F);
		xfer__SOURCE_BOX__ (F);
		xfer__X_LIST__ (F);
		xfer__SP_DC__ (F);
		xfer__RADIUS__ (F);
		xfer__CENTER__ (F);
		xfer__TRG_DWN_CHK_VAL__ (F);
	#endif
  t_x_start = stopwatch_elapsed (timer);
	xlist_calc__gpu (F);
	hipDeviceSynchronize ();
  t_x = stopwatch_elapsed (timer);
	#if MIN_DATA
		/* backup target box */
		xfer__TARGET_BOX__back (F);
		/* backup TRG_DWN_CHK_VAL */
		xfer__TRG_DWN_CHK_VAL__back (F);

		free__TAG__ (F);
		free__TRGNUM__ (F);
		free__CHILDREN__ (F);
		free__TARGET_BOX__ (F);
		free__SOURCE_BOX__ (F);
		free__X_LIST__ (F);
		free__SP_DC__ (F);
		free__RADIUS__ (F);
		free__CENTER__ (F);
		free__TRG_DWN_CHK_VAL__ (F);
	#endif	
  fprintf (stderr, "Done Xlist.\n ");

  fprintf (stderr, "Performing Down calculation ...\n");
	#if MIN_DATA
		alloc__TAG__ (F);
		alloc__TRG_DWN_CHK_VAL__ (F);
		alloc__DC2DE__ (F);
		alloc__TRG_DWN_EQU_DEN__ (F);
		alloc__DEPTH__ (F);
		alloc__PATH2NODE__ (F);
		alloc__PARENT__ (F);
		alloc__DE2DC__ (F);
		alloc__TARGET_BOX__ (F);
		alloc__SP_DE__ (F);
		alloc__RADIUS__ (F);
		alloc__CENTER__ (F);

		xfer__TAG__ (F);
		xfer__TRG_DWN_CHK_VAL__ (F);
		xfer__DC2DE__ (F);
		xfer__DEPTH__ (F);
		xfer__PATH2NODE__ (F);
		xfer__PARENT__ (F);
		xfer__DE2DC__ (F);
		xfer__TARGET_BOX__ (F);
		xfer__SP_DE__ (F);
		xfer__RADIUS__ (F);
		xfer__CENTER__ (F);
	#endif
  t_down_start = stopwatch_elapsed (timer);
  down_calc__gpu (F);
	hipDeviceSynchronize ();
  t_down = stopwatch_elapsed (timer);
	#if MIN_DATA
		free__TAG__ (F);
		free__TRG_DWN_CHK_VAL__ (F);
		free__DC2DE__ (F);
		free__TRG_DWN_EQU_DEN__ (F);
		free__DEPTH__ (F);
		free__PATH2NODE__ (F);
		free__PARENT__ (F);
		free__DE2DC__ (F);
		free__SP_DE__ (F);
		free__RADIUS__ (F);
		free__CENTER__ (F);

		// do this after copy_trg_val__gpu
		// free__TARGET_BOX__ (F);
	#endif
  fprintf (stderr, "Done Down.\n");

  t_copy_start = stopwatch_stop (timer);
  copy_trg_val__gpu (F);
  t_copy = stopwatch_stop (timer);

	t_v = (t_v1 - t_v1_start) + (t_v2 - t_v2_start) + (t_v3 - t_v3_start) + (t_v4 - t_v4_start);
	t_total = t_up + (t_u - t_u_start) + (t_v) + (t_w - t_w_start) + (t_x - t_x_start) + (t_down - t_down_start) + (t_copy - t_copy_start);

	#if MIN_DATA
		free__TARGET_BOX__ (F);
	#endif
  
  fprintf(stderr, "=== Statistics ===\n");
  fprintf (stdout, "Code: %s\n", get_implementation_name ());
  fprintf (stdout, "Floating-point word size: %lu bytes\n", sizeof (real_t));
  fprintf (stderr, "== Timing for FMM3d ==\n");
  fprintf (stderr, "  Up    : %Lg secs (%.1Lf%%)\n", t_up, t_up / t_total * 100);
  fprintf (stderr, "  U list: %Lg secs (%.1Lf%%)\n", (t_u - t_u_start), (t_u - t_u_start) / t_total * 100);
  fprintf (stderr, "  V list: %Lg secs (%.1Lf%%)\n", t_v, t_v / t_total * 100);
  fprintf (stderr, "  W list: %Lg secs (%.1Lf%%)\n", (t_w - t_w_start), (t_w - t_w_start) / t_total * 100);
  fprintf (stderr, "  X list: %Lg secs (%.1Lf%%)\n", (t_x - t_x_start), (t_x - t_x_start) / t_total * 100);
  fprintf (stderr, "  Down  : %Lg secs (%.1Lf%%)\n", (t_down - t_down_start), (t_down - t_down_start) / t_total * 100);
  fprintf (stderr, "  ==> Total Execution Time: %Lg secs\n", t_total);

  stopwatch_destroy (timer);

  return 0;
}
/* ------------------------------------------------------------------------
 */
