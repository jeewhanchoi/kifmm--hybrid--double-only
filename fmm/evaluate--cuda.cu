#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#include "evaluate.h"
#include "util.h"
#include "reals.h"
#include "partial.h"
#include "../timing/timing.h"
#include "node_gpu.h"
#include <cutil_inline.h>

/* ------------------------------------------------------------------------
 */
int
get_byte_padding()
{
	return getenv__int("BYTEPAD", 128);
}

/* ------------------------------------------------------------------------
 */
void
xlist_create_xlist (UList_t* U, int num_boxes, AllNodes* All_N)
{
	int i, j, nu;
	int list_size = 0;
	
	assert (U && All_N);

	vector<NodeTree>& nodeVec = *All_N->N;

	/* allocate memory for ulist ptr */
	U->n_boxes_ = num_boxes;
	U->Ptr_ = (int *) malloc (sizeof (int) * (num_boxes + 1));
	assert (U->Ptr_);

	/* See how big ulist should be */
	U->Ptr_[0] = 0;
	for(i = 0; i < U->n_boxes_; i++) {
		list_size += nodeVec[i].Xnodes.size ();
		U->Ptr_[i + 1] = list_size;
	}

	/* allocate memory for ulist */
	U->L_ = (int*) malloc (sizeof (int) * list_size);
	assert (U->L_);

	/* initialize ulist */
	for(i = 0; i < U->n_boxes_; i++) {
		nu = nodeVec[i].Xnodes.size ();
		for(j = 0; j < nu; j++) {
			U->L_[U->Ptr_[i] + j] = nodeVec[i].Xnodes[j];
		}
	}
}




void
wlist_create_wlist (UList_t* U, int num_boxes, AllNodes* All_N)
{
	int i, j, nu;
	int list_size = 0;
	
	assert (U && All_N);

	vector<NodeTree>& nodeVec = *All_N->N;

	/* allocate memory for ulist ptr */
	U->n_boxes_ = num_boxes;
	U->Ptr_ = (int *) malloc (sizeof (int) * (num_boxes + 1));
	assert (U->Ptr_);

	/* See how big ulist should be */
	U->Ptr_[0] = 0;
	for(i = 0; i < U->n_boxes_; i++) {
		list_size += nodeVec[i].Wnodes.size ();
		U->Ptr_[i + 1] = list_size;
	}

	/* allocate memory for ulist */
	U->L_ = (int*) malloc (sizeof (int) * list_size);
	assert (U->L_);

	/* initialize ulist */
	for(i = 0; i < U->n_boxes_; i++) {
		nu = nodeVec[i].Wnodes.size ();
		for(j = 0; j < nu; j++) {
			U->L_[U->Ptr_[i] + j] = nodeVec[i].Wnodes[j];
		}
	}
}


void
ulist_create_ulist (UList_t* U, int num_boxes, AllNodes* All_N)
{
	int i, j, nu;
	int list_size = 0;
	
	assert (U && All_N);

	vector<NodeTree>& nodeVec = *All_N->N;

	/* allocate memory for ulist ptr */
	U->n_boxes_ = num_boxes;
	U->Ptr_ = (int *) malloc (sizeof (int) * (num_boxes + 1));
	assert (U->Ptr_);

	/* See how big ulist should be */
	U->Ptr_[0] = 0;
	for(i = 0; i < U->n_boxes_; i++) {
		list_size += nodeVec[i].Unodes.size ();
		U->Ptr_[i + 1] = list_size;
	}

	/* allocate memory for ulist */
	U->L_ = (int*) malloc (sizeof (int) * list_size);
	assert (U->L_);

	/* initialize ulist */
	for(i = 0; i < U->n_boxes_; i++) {
		nu = nodeVec[i].Unodes.size ();
		for(j = 0; j < nu; j++) {
			U->L_[U->Ptr_[i] + j] = nodeVec[i].Unodes[j];
		}
	}
}

/* ------------------------------------------------------------------------
 */

void
ulist_create_boxes__double_source (AllNodes *All_N, FMMWrapper_t *F)
{
	int i, j, n;
	int padding, n_padded, n_points_, n_points_padded_;

	vector<NodeTree>& nodeVec = *All_N->N;

	Boxes_t *B;
	Node *N;

	B = &F->S_h_;
	N = All_N->Ns;

	assert (B && N);

	padding = get_byte_padding () / sizeof (dtype);

	B->n_boxes_ = nodeVec.size ();
	B->Bptr_ = (int *) malloc (sizeof (int) * (B->n_boxes_ + 1));
	B->Bn_ = (int *) malloc (sizeof (int) * B->n_boxes_);
	assert (B->Bptr_ && B->Bn_);

	n_points_ = 0;
	n_points_padded_ = 0;
	B->Bptr_[0] = 0;
	
	for(i = 0; i < B->n_boxes_; i++) {
		if(nodeVec[i].tag & LET_SRCNODE && nodeVec[i].child == -1) {
			n = N[i].num_pts;
			n_padded = ((n + padding - 1) / padding) * padding;
			assert (n_padded >= n);

			B->Bn_[i] = n;
			B->Bptr_[i + 1] = B->Bptr_[i] + n_padded;

			n_points_ += n;
			n_points_padded_ += n_padded;
		} else {
			B->Bn_[i] = 0;
			B->Bptr_[i + 1] = B->Bptr_[i];
		}
	}
	assert (n_points_padded_ == B->Bptr_[B->n_boxes_]);

	B->n_points_ = n_points_;
	B->x_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	B->y_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	B->z_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	B->w_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	assert (B->x_ && B->y_ && B->z_ && B->w_);

	/* copy points */
	for(i = 0; i < B->n_boxes_; i++) {
		if(nodeVec[i].tag & LET_SRCNODE) {
			n = N[i].num_pts;
			for(j = 0; j < n; j++) {
				B->x_[B->Bptr_[i] + j] = N[i].x[j];
				B->y_[B->Bptr_[i] + j] = N[i].y[j];
				B->z_[B->Bptr_[i] + j] = N[i].z[j];
				B->w_[B->Bptr_[i] + j] = N[i].den_pot[j];
			}
		}
	}

}


void
ulist_create_boxes__double_target (AllNodes *All_N, FMMWrapper_t *F)
{
	int i, j, n;
	int padding, n_padded, n_points_, n_points_padded_;

	vector<NodeTree>& nodeVec = *All_N->N;

	Boxes_t *B;
	Node *N;

	B = &F->T_h_;
	N = All_N->Nt;

	assert (B && N);

	padding = get_byte_padding () / sizeof (dtype);

	B->n_boxes_ = nodeVec.size ();
	B->Bptr_ = (int *) malloc (sizeof (int) * (B->n_boxes_ + 1));
	B->Bn_ = (int *) malloc (sizeof (int) * B->n_boxes_);
	assert (B->Bptr_ && B->Bn_);

	n_points_ = 0;
	n_points_padded_ = 0;
	B->Bptr_[0] = 0;
	
	for(i = 0; i < B->n_boxes_; i++) {
		if(nodeVec[i].tag & LET_TRGNODE && nodeVec[i].child == -1) {
			n = N[i].num_pts;
			n_padded = ((n + padding - 1) / padding) * padding;
			assert (n_padded >= n);

			B->Bn_[i] = n;
			B->Bptr_[i + 1] = B->Bptr_[i] + n_padded;

			n_points_ += n;
			n_points_padded_ += n_padded;
		} else {
			B->Bn_[i] = 0;
			B->Bptr_[i + 1] = B->Bptr_[i];
		}
	}
	assert (n_points_padded_ == B->Bptr_[B->n_boxes_]);

	B->n_points_ = n_points_;
	B->x_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	B->y_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	B->z_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	B->w_ = (real_t *) malloc (n_points_padded_ * sizeof (real_t));
	assert (B->x_ && B->y_ && B->z_ && B->w_);

	/* copy points */
	for(i = 0; i < B->n_boxes_; i++) {
		if(nodeVec[i].tag & LET_TRGNODE) {
			n = N[i].num_pts;
			for(j = 0; j < n; j++) {
				B->x_[B->Bptr_[i] + j] = N[i].x[j];
				B->y_[B->Bptr_[i] + j] = N[i].y[j];
				B->z_[B->Bptr_[i] + j] = N[i].z[j];
				B->w_[B->Bptr_[i] + j] = N[i].den_pot[j];
			}
		}
	}
}

void
ulist_create_boxes__double (Boxes_t* B, 
													 int num_boxes, 
													 const Node* N, 
													 int padding)
{
	int i, j, n, n_padded, min;

	/* total number of points */
	int n_points_ = 0;
	int n_points_padded_ = 0;

	/* check if structures that were passed in are valid */
	assert (B && N);

	/* allocate memory to data structures */
	B->n_boxes_ = num_boxes;
	B->Bptr_ = (int *) malloc (sizeof (int) * (B->n_boxes_ + 1));
	B->Bn_ = (int *) malloc (sizeof (int) * B->n_boxes_);
	assert (B->Bptr_ && B->Bn_);

	/* initialize data structures */
	B->Bptr_[0] = 0;
	for(i = 0; i < num_boxes; i++) {
		/* number of points in this box */
		n = N[i].num_pts;
		/* number of points in this box if padded */
		n_padded = ((n + padding - 1) / padding) * padding;
		assert (n_padded >= n);

		/* make Bn_ and Bptr_ have/point to the right values */
		B->Bn_[i] = n;
		B->Bptr_[i+1] = B->Bptr_[i] + n_padded;

		n_points_ += n;
		n_points_padded_ += n_padded;
	}
	assert (n_points_padded_ == B->Bptr_[B->n_boxes_]);

	/* allocate memory to data structures that are going to hold the values */
	B->n_points_ = n_points_;	
	B->x_ = (real_t*) malloc (n_points_padded_ * sizeof (real_t));
	B->y_ = (real_t*) malloc (n_points_padded_ * sizeof (real_t));
	B->z_ = (real_t*) malloc (n_points_padded_ * sizeof (real_t));
	B->w_ = (real_t*) malloc (n_points_padded_ * sizeof (real_t));
	assert (B->x_ && B->y_ && B->z_ && B->w_);

	/* copy points */
	for(i = 0; i < num_boxes; i++) {
		n = N[i].num_pts;
		min = B->Bptr_[i];
		for(j = 0; j < n; j++) {
			B->x_[min + j] = N[i].x[j];
			B->y_[min + j] = N[i].y[j];
			B->z_[min + j] = N[i].z[j];
			B->w_[min + j] = N[i].den_pot[j];
		}
	}
}

/* ------------------------------------------------------------------------
 */

void
alloc__SOURCE_BOX__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* Source boxes */
	f->S_d_.n_points_ = f->S_h_.n_points_;
	f->S_d_.n_boxes_ = f->S_h_.n_boxes_;
	/* Allocate memory for data */
	cutilSafeCall (hipMalloc ((void**)&f->S_d_.x_, 
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] 
														 * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->S_d_.y_, 
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] 
														 * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->S_d_.z_, 
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] 
														 * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->S_d_.w_, 
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] 
														 * sizeof (dtype)));
	/* Allocate memory for pointers */
	cutilSafeCall (hipMalloc ((void**)&f->S_d_.Bptr_,
														 (f->S_d_.n_boxes_ + 1) * sizeof (int)));
	cutilSafeCall (hipMalloc ((void**)&f->S_d_.Bn_,
														 f->S_d_.n_boxes_ * sizeof (int)));
	assert (&f->S_d_ && &f->S_h_);
  /* ------------------------------------------------------------ */
}

void
alloc__TARGET_BOX__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* Target boxes */
	f->T_d_.n_points_ = f->T_h_.n_points_;
	f->T_d_.n_boxes_ = f->T_h_.n_boxes_;
	
	/* Allocate memory for data */
	cutilSafeCall (hipMalloc ((void**)&f->T_d_.x_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] 
														 * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->T_d_.y_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] 
														 * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->T_d_.z_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] 
														 * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->T_d_.w_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] 
														 * sizeof (dtype)));
	/* Allocate memory for pointers */
	cutilSafeCall (hipMalloc ((void**)&f->T_d_.Bptr_,
														 (f->T_h_.n_boxes_ + 1) * sizeof (int)));
	cutilSafeCall (hipMalloc ((void**)&f->T_d_.Bn_,
														 f->T_h_.n_boxes_ * sizeof (int)));
	assert (&f->T_d_ && &f->T_h_);
  /* ------------------------------------------------------------ */
}

void
alloc__U_LIST__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* ulist */
	f->U_d_.n_boxes_ = f->U_h_.n_boxes_;
	cutilSafeCall (hipMalloc ((void**)&f->U_d_.L_,
														 f->U_h_.Ptr_[f->U_h_.n_boxes_] * sizeof (int)));
	cutilSafeCall (hipMalloc ((void**)&f->U_d_.Ptr_,
														 (f->U_h_.n_boxes_ + 1) * sizeof (int)));
	assert (&f->U_d_ && &f->U_h_);
  /* ------------------------------------------------------------ */
}

void
alloc__TAG__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* tag */
	cutilSafeCall (hipMalloc ((void**)&f->tag_d_, 
														 nodeVec.size () * sizeof (int)));
	assert (f->tag_d_);
  /* ------------------------------------------------------------ */
}


void
alloc__DEPTH__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* depth */
	cutilSafeCall (hipMalloc ((void**)&f->depth_d_, 
														 nodeVec.size () * sizeof (int)));
	assert (f->depth_d_);
  /* ------------------------------------------------------------ */
}


void
alloc__CHILDREN__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* children */
	/*
	cutilSafeCall (hipMalloc ((void**)&f->child_d_, 
														 num_non_leaf_nodes * sizeof (int)));
	 */
	cutilSafeCall (hipMalloc ((void**)&f->child_d_, 
														 nodeVec.size () * sizeof (int)));
	assert (f->child_d_);
  /* ------------------------------------------------------------ */
}


void
alloc__RADIUS__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* radius */
	/*
	cutilSafeCall (hipMalloc ((void**)&f->radius_d_,
														 num_leaf_nodes * sizeof (dtype)));
	 */
	cutilSafeCall (hipMalloc ((void**)&f->radius_d_,
														 nodeVec.size () * sizeof (dtype)));
	assert (f->radius_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__CENTER__ (FMMWrapper_t* f) {
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* radius */
	/*
	cutilSafeCall (hipMalloc ((void**)&f->radius_d_,
														 num_leaf_nodes * sizeof (dtype)));
	 */
	cutilSafeCall (hipMalloc ((void**)&f->radius_d_,
														 nodeVec.size () * sizeof (dtype)));
  /* ------------------------------------------------------------ */
	/* center */
	cutilSafeCall (hipMalloc ((void**)&f->center0_d_,
														 nodeVec.size () * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->center1_d_,
														 nodeVec.size () * sizeof (dtype)));
	cutilSafeCall (hipMalloc ((void**)&f->center2_d_,
														 nodeVec.size () * sizeof (dtype)));
	assert (f->center0_d_ && f->center1_d_ && f->center2_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__SP_UC__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* SP[UC] */
	cutilSafeCall (hipMalloc ((void**)&f->SP_UC_d_,
														 3 * f->SP_UC_size_padded * sizeof (dtype)));
	assert (f->SP_UC_d_);
  /* ------------------------------------------------------------ */
}


void
alloc__UC2UE__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* UC2UE matrix */
	cutilSafeCall (hipMalloc ((void**)&f->UC2UE_d_,
														 f->UC2UE_r_padded * f->UC2UE_c * sizeof (dtype)));
	assert (f->UC2UE_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__UE2UC__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* UE2UC matrix */
	cutilSafeCall (hipMalloc ((void**)&f->UE2UC_d_,
														 (2 * 2 * 2) * (f->UE2UC_r_padded * f->UE2UC_c) *
														 sizeof (dtype)));
	assert (f->UE2UC_d_);
  /* ------------------------------------------------------------ */
}


void
alloc__SRC_UPW_EQU_DEN__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* Temporary up_calc GPU variables */
	/* src_upw_equ_den */
	cutilSafeCall (hipMalloc ((void**)&f->SRC_UPW_EQU_DEN_d_,
														 nodeVec.size () * f->UC2UE_r_padded * 
														 sizeof (dtype)));
	assert (f->SRC_UPW_EQU_DEN_d_);
  /* ------------------------------------------------------------ */

}


void
alloc__VLIST_SRC__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* src */
	cutilSafeCall (hipMalloc ((void**)&f->vlist_src_d_,
														 nodeVec.size () * f->vlist_array_size *
														 sizeof (dtype)));
	assert (f->vlist_src_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__REG_DEN__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
  /* reg_den */
	cutilSafeCall (hipMalloc ((void**)&f->reg_den_d_,
														 nodeVec.size () * f->reg_den_size *
														 sizeof (dtype)));
	assert (f->reg_den_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__TT__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* tt */
	cutilSafeCall (hipMalloc ((void**)&f->tt, f->trans_arrays_num * f->RP_n_ * 	
														 sizeof (dtype)));
	assert (f->tt);
  /* ------------------------------------------------------------ */
}

void
alloc__VLIST_TRANS__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
  /* trans */
	cutilSafeCall (hipMalloc ((void**)&f->vlist_trans_d_,
														 f->trans_arrays_num * f->vlist_array_size *
														 sizeof (dtype)));
  assert (f->vlist_trans_d_);
  /* ------------------------------------------------------------ */
}


void
alloc__VLIST_TRG__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* trg */
	cutilSafeCall (hipMalloc ((void**)&f->vlist_trg_d_,
														 nodeVec.size () * f->vlist_array_size * 
														 sizeof (dtype)));
	assert (f->vlist_trg_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__VLIST_TLIST__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* vlist and tlist and pointer */
	/* pointer */
	cutilSafeCall (hipMalloc ((void**)&f->vlist_ptr_d_, 
														 (nodeVec.size () + 1) * sizeof (int)));
	assert (f->vlist_ptr_d_);
	/* vlist */
	cutilSafeCall (hipMalloc ((void**)&f->vlist_d_, f->list_size * sizeof (int)));
	assert (f->vlist_d_);
	/* tlist */
	cutilSafeCall (hipMalloc ((void**)&f->tlist_d_, f->list_size * sizeof (int)));
	assert (f->tlist_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__TRG_DWN_CHK_VAL__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
  /* trg_dwn_chk_val */
	cutilSafeCall (hipMalloc ((void**)&f->TRG_DWN_CHK_VAL_d_,
														 nodeVec.size () * f->SP_DC_n_padded_ * 
														 sizeof (dtype)));
	assert (f->TRG_DWN_CHK_VAL_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__PATH2NODE__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* path2Node */
	cutilSafeCall (hipMalloc ((void**)&f->path2Node_d_,
														 nodeVec.size () * sizeof (int3)));
	assert (f->path2Node_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__PARENT__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* parent */
	/* Not needed - use children structure instead */
	/* Now I need it */
	cutilSafeCall (hipMalloc ((void**)&f->parent_d_, 
														 nodeVec.size () * sizeof (int)));
	assert (f->parent_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__TRG_DWN_EQU_DEN__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  /* ------------------------------------------------------------ */
	/* trg_dwn_equ_den */
	cutilSafeCall (hipMalloc ((void**)&f->TRG_DWN_EQU_DEN_d_,
														 nodeVec.size () * f->SP_DE_n_padded * 
														 sizeof (dtype)));
	assert (f->TRG_DWN_EQU_DEN_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__DC2DE__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* DC2DE_mat */
	cutilSafeCall (hipMalloc ((void**)&f->DC2DE_d_,
														 f->DC2DE_r_padded * f->DC2DE_c * sizeof (dtype)));
	assert (f->DC2DE_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__DE2DC__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* DE2DC_mat[8] */
	cutilSafeCall (hipMalloc ((void**)&f->DE2DC_d_,
														 (2 * 2 * 2) * f->DE2DC_r_padded * f->DE2DC_c *
														 sizeof (dtype)));
	assert (f->DE2DC_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__SP_DE__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* SP[DE] */
	cutilSafeCall (hipMalloc ((void**)&f->SP_DE_d_,
														 3 * f->SP_DE_n_padded * sizeof (dtype)));
	assert (f->SP_DE_d_);
  /* ------------------------------------------------------------ */
}

void
alloc__SP_UE__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMalloc ((void**)&f->SP_UE_d_,
														 3 * f->SP_UE_n_padded * sizeof (dtype)));
	assert (f->SP_UE_d_);
}

void
alloc__W_LIST__ (FMMWrapper_t* f) 
{
  /* ------------------------------------------------------------ */
	/* ulist */
	f->W_d_.n_boxes_ = f->W_h_.n_boxes_;
	cutilSafeCall (hipMalloc ((void**)&f->W_d_.L_,
														 f->W_h_.Ptr_[f->W_h_.n_boxes_] * sizeof (int)));
	cutilSafeCall (hipMalloc ((void**)&f->W_d_.Ptr_,
														 (f->W_h_.n_boxes_ + 1) * sizeof (int)));
	assert (&f->W_d_ && &f->W_h_);
  /* ------------------------------------------------------------ */
}

void
alloc__SRCNUM__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMalloc ((void**)&f->srcNum_d_,
														 nodeVec.size () * sizeof (int)));
	assert (f->srcNum_d_);
}

void
alloc__SP_DC__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMalloc ((void**)&f->SP_DC_d_,
														 3 * f->SP_DC_n_padded_ * sizeof (dtype)));
	assert (f->SP_DC_d_);
}

void
alloc__X_LIST__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* ulist */
	f->X_d_.n_boxes_ = f->X_h_.n_boxes_;
	cutilSafeCall (hipMalloc ((void**)&f->X_d_.L_,
														 f->X_h_.Ptr_[f->X_h_.n_boxes_] * sizeof (int)));
	cutilSafeCall (hipMalloc ((void**)&f->X_d_.Ptr_,
														 (f->X_h_.n_boxes_ + 1) * sizeof (int)));
	assert (&f->X_d_ && &f->X_h_);
  /* ------------------------------------------------------------ */
}

void
alloc__TRGNUM__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMalloc ((void**)&f->trgNum_d_,
														 nodeVec.size () * sizeof (int)));
	assert (f->trgNum_d_);
}


void
xfer__SOURCE_BOX__ (FMMWrapper_t* f)
{
	/* Source boxes */
	cutilSafeCall (hipMemcpy (f->S_d_.x_, f->S_h_.x_,
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->S_d_.y_, f->S_h_.y_,
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->S_d_.z_, f->S_h_.z_,
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->S_d_.w_, f->S_h_.w_,
														 f->S_h_.Bptr_[f->S_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->S_d_.Bptr_, f->S_h_.Bptr_,
														 (f->S_h_.n_boxes_ + 1) * sizeof (int),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->S_d_.Bn_, f->S_h_.Bn_,
														 f->S_h_.n_boxes_ * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__TARGET_BOX__ (FMMWrapper_t* f)
{
	/* Target boxes */
	cutilSafeCall (hipMemcpy (f->T_d_.x_, f->T_h_.x_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->T_d_.y_, f->T_h_.y_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->T_d_.z_, f->T_h_.z_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->T_d_.w_, f->T_h_.w_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->T_d_.Bptr_, f->T_h_.Bptr_,
														 (f->T_h_.n_boxes_ + 1) * sizeof (int),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->T_d_.Bn_, f->T_h_.Bn_,
														 f->T_h_.n_boxes_ * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__U_LIST__ (FMMWrapper_t* f)
{
	/* Ulist */
	cutilSafeCall (hipMemcpy (f->U_d_.L_, f->U_h_.L_,
														 f->U_h_.Ptr_[f->U_h_.n_boxes_] * 
														 sizeof (int), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->U_d_.Ptr_, f->U_h_.Ptr_,
														 (f->U_h_.n_boxes_ + 1) * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__TAG__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->tag_d_, f->tag_h_,
														 nodeVec.size () * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__DEPTH__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	/* depth */
	cutilSafeCall (hipMemcpy (f->depth_d_, f->depth_h_,
														 nodeVec.size () * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__CHILDREN__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->child_d_, f->child_h_,
														 nodeVec.size () * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__RADIUS__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->radius_d_, f->radius_h_,
														 nodeVec.size () * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__CENTER__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->center0_d_, f->center0_h_,
														 nodeVec.size () * sizeof (dtype),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->center1_d_, f->center1_h_,
														 nodeVec.size () * sizeof (dtype),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->center2_d_, f->center2_h_,
														 nodeVec.size () * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__SP_UC__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->SP_UC_d_, f->SP_UC_h_,
														 3 * f->SP_UC_size_padded * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__UC2UE__ (FMMWrapper_t* f)
{
	/* UC2UE matrix */
	cutilSafeCall (hipMemcpy (f->UC2UE_d_, f->UC2UE_h_,
														 f->UC2UE_r_padded * f->UC2UE_c * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__UE2UC__ (FMMWrapper_t* f)
{
	/* UE2UC matrix */
	cutilSafeCall (hipMemcpy (f->UE2UC_d_, f->UE2UC_h_,
														 (2 * 2 * 2) * (f->UE2UC_r_padded * f->UE2UC_c) * 
														 sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__SRC_UPW_EQU_DEN__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->SRC_UPW_EQU_DEN_d_, f->SRC_UPW_EQU_DEN_h_,
														 nodeVec.size () * f->UC2UE_r_padded *
														 sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__VLIST_TLIST__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->vlist_d_, f->vlist_h_,
														 f->list_size * sizeof (int),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->tlist_d_, f->tlist_h_,
														 f->list_size * sizeof (int),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->vlist_ptr_d_, f->vlist_ptr_h_,
														 (nodeVec.size () + 1) * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__TRG_DWN_CHK_VAL__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->TRG_DWN_CHK_VAL_d_, f->TRG_DWN_CHK_VAL_h_,
														 nodeVec.size () * f->SP_DC_n_padded_ *
														 sizeof (dtype),
														 hipMemcpyHostToDevice));
}


void
xfer__PATH2NODE__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->path2Node_d_, f->path2Node_h_,
														 nodeVec.size () * sizeof (int3),
														 hipMemcpyHostToDevice));
}

void
xfer__PARENT__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->parent_d_, f->parent_h_,
														 nodeVec.size () * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__DC2DE__(FMMWrapper_t* f)
{
	/* DC2DE_mat */
	cutilSafeCall (hipMemcpy (f->DC2DE_d_, f->DC2DE_h_,
														 f->DC2DE_r_padded * f->DC2DE_c * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__DE2DC__(FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->DE2DC_d_, f->DE2DC_h_,
														 (2 * 2 * 2) * f->DE2DC_r_padded * f->DE2DC_c *
														 sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__SP_DE__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->SP_DE_d_, f->SP_DE_h_,
														 3 * f->SP_DE_n_padded * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__SP_UE__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->SP_UE_d_, f->SP_UE_h_,
														 3 * f->SP_UE_n_padded * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__W_LIST__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->W_d_.L_, f->W_h_.L_,
														 f->W_h_.Ptr_[f->W_h_.n_boxes_] * 
														 sizeof (int), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->W_d_.Ptr_, f->W_h_.Ptr_,
														 (f->W_h_.n_boxes_ + 1) * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__SRCNUM__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->srcNum_d_, f->srcNum_h_,
														 nodeVec.size () * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__SP_DC__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->SP_DC_d_, f->SP_DC_h_,
														 3 * f->SP_DC_n_padded_ * sizeof (dtype),
														 hipMemcpyHostToDevice));
}

void
xfer__X_LIST__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->X_d_.L_, f->X_h_.L_,
														 f->X_h_.Ptr_[f->X_h_.n_boxes_] * 
														 sizeof (int), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (f->X_d_.Ptr_, f->X_h_.Ptr_,
														 (f->X_h_.n_boxes_ + 1) * sizeof (int),
														 hipMemcpyHostToDevice));
}

void
xfer__TRGNUM__ (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->trgNum_d_, f->trgNum_h_,
														 nodeVec.size () * sizeof (int),
														 hipMemcpyHostToDevice));
}


void
xfer__SRC_UPW_EQU_DEN__back (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->SRC_UPW_EQU_DEN_h_, f->SRC_UPW_EQU_DEN_d_,
														 nodeVec.size () * f->UC2UE_r_padded *
														 sizeof (dtype),
														 hipMemcpyDeviceToHost));
}

void
xfer__TRG_DWN_CHK_VAL__back (FMMWrapper_t* f)
{
  AllNodes *All_N = f->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

	cutilSafeCall (hipMemcpy (f->TRG_DWN_CHK_VAL_h_, f->TRG_DWN_CHK_VAL_d_,
														 nodeVec.size () * f->SP_DC_n_padded_ *
														 sizeof (dtype),
														 hipMemcpyDeviceToHost));
}


void
xfer__TARGET_BOX__back (FMMWrapper_t* f)
{
	cutilSafeCall (hipMemcpy (f->T_h_.w_, f->T_d_.w_,
														 f->T_h_.Bptr_[f->T_h_.n_boxes_] * 
														 sizeof (dtype), hipMemcpyDeviceToHost));
}


void
free__SOURCE_BOX__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* Deallocate memory for data */
	cutilSafeCall (hipFree (f->S_d_.x_));
	cutilSafeCall (hipFree (f->S_d_.y_));
	cutilSafeCall (hipFree (f->S_d_.z_));
	cutilSafeCall (hipFree (f->S_d_.w_));
	/* Deallocate memory for pointers */
	cutilSafeCall (hipFree (f->S_d_.Bptr_));
	cutilSafeCall (hipFree (f->S_d_.Bn_));
  /* ------------------------------------------------------------ */
}

void
free__TARGET_BOX__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* Target boxes */
	cutilSafeCall (hipFree (f->T_d_.x_));
	cutilSafeCall (hipFree (f->T_d_.y_));
	cutilSafeCall (hipFree (f->T_d_.z_));
	cutilSafeCall (hipFree (f->T_d_.w_));
	/* Allocate memory for pointers */
	cutilSafeCall (hipFree (f->T_d_.Bptr_));
	cutilSafeCall (hipFree (f->T_d_.Bn_));
  /* ------------------------------------------------------------ */
}

void
free__U_LIST__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* ulist */
	cutilSafeCall (hipFree (f->U_d_.L_));
	cutilSafeCall (hipFree (f->U_d_.Ptr_));
  /* ------------------------------------------------------------ */
}

void
free__TAG__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* tag */
	cutilSafeCall (hipFree (f->tag_d_));
  /* ------------------------------------------------------------ */
}


void
free__DEPTH__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* depth */
	cutilSafeCall (hipFree (f->depth_d_));
  /* ------------------------------------------------------------ */
}


void
free__CHILDREN__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* children */
	cutilSafeCall (hipFree (f->child_d_));
  /* ------------------------------------------------------------ */
}


void
free__RADIUS__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* radius */
	cutilSafeCall (hipFree (f->radius_d_));
  /* ------------------------------------------------------------ */
}

void
free__CENTER__ (FMMWrapper_t* f) {
	/* ------------------------------------------------------------ */
	cutilSafeCall (hipFree (f->center0_d_));
	cutilSafeCall (hipFree (f->center1_d_));
	cutilSafeCall (hipFree (f->center2_d_));
  /* ------------------------------------------------------------ */
}

void
free__SP_UC__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* SP[UC] */
	cutilSafeCall (hipFree (f->SP_UC_d_));
  /* ------------------------------------------------------------ */
}


void
free__UC2UE__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* UC2UE matrix */
	cutilSafeCall (hipFree (f->UC2UE_d_));
  /* ------------------------------------------------------------ */
}

void
free__UE2UC__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* UE2UC matrix */
	cutilSafeCall (hipFree (f->UE2UC_d_));
  /* ------------------------------------------------------------ */
}


void
free__SRC_UPW_EQU_DEN__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* Temporary up_calc GPU variables */
	/* src_upw_equ_den */
	cutilSafeCall (hipFree (f->SRC_UPW_EQU_DEN_d_));
  /* ------------------------------------------------------------ */

}


void
free__VLIST_SRC__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* src */
	cutilSafeCall (hipFree (f->vlist_src_d_));
  /* ------------------------------------------------------------ */
}

void
free__REG_DEN__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
  /* reg_den */
	cutilSafeCall (hipFree (f->reg_den_d_));
  /* ------------------------------------------------------------ */
}

void
free__TT__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* tt */
	cutilSafeCall (hipFree (f->tt));
  /* ------------------------------------------------------------ */
}

void
free__VLIST_TRANS__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
  /* trans */
	cutilSafeCall (hipFree (f->vlist_trans_d_));
  /* ------------------------------------------------------------ */
}


void
free__VLIST_TRG__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* trg */
	cutilSafeCall (hipFree (f->vlist_trg_d_));
  /* ------------------------------------------------------------ */
}

void
free__VLIST_TLIST__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* vlist and tlist and pointer */
	/* pointer */
	cutilSafeCall (hipFree (f->vlist_ptr_d_));
	/* vlist */
	cutilSafeCall (hipFree (f->vlist_d_));
	/* tlist */
	cutilSafeCall (hipFree (f->tlist_d_));
  /* ------------------------------------------------------------ */
}

void
free__TRG_DWN_CHK_VAL__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
  /* trg_dwn_chk_val */
	cutilSafeCall (hipFree (f->TRG_DWN_CHK_VAL_d_));
  /* ------------------------------------------------------------ */
}

void
free__PATH2NODE__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* path2Node */
	cutilSafeCall (hipFree (f->path2Node_d_));
  /* ------------------------------------------------------------ */
}

void
free__PARENT__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* parent */
	/* Not needed - use children structure instead */
	/* Now I need it */
	cutilSafeCall (hipFree (f->parent_d_));
  /* ------------------------------------------------------------ */
}

void
free__TRG_DWN_EQU_DEN__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* trg_dwn_equ_den */
	cutilSafeCall (hipFree (f->TRG_DWN_EQU_DEN_d_));
  /* ------------------------------------------------------------ */
}

void
free__DC2DE__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* DC2DE_mat */
	cutilSafeCall (hipFree (f->DC2DE_d_));
  /* ------------------------------------------------------------ */
}

void
free__DE2DC__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* DE2DC_mat[8] */
	cutilSafeCall (hipFree (f->DE2DC_d_));
  /* ------------------------------------------------------------ */
}

void
free__SP_DE__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* SP[DE] */
	cutilSafeCall (hipFree (f->SP_DE_d_));
  /* ------------------------------------------------------------ */
}

void
free__SP_UE__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipFree (f->SP_UE_d_));
}

void
free__W_LIST__ (FMMWrapper_t* f) 
{
  /* ------------------------------------------------------------ */
	/* ulist */
	cutilSafeCall (hipFree (f->W_d_.L_));
	cutilSafeCall (hipFree (f->W_d_.Ptr_));
  /* ------------------------------------------------------------ */
}

void
free__SRCNUM__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipFree (f->srcNum_d_));
}

void
free__SP_DC__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipFree (f->SP_DC_d_));
}

void
free__X_LIST__ (FMMWrapper_t* f)
{
  /* ------------------------------------------------------------ */
	/* ulist */
	cutilSafeCall (hipFree (f->X_d_.L_));
	cutilSafeCall (hipFree (f->X_d_.Ptr_));
  /* ------------------------------------------------------------ */
}

void
free__TRGNUM__ (FMMWrapper_t* f)
{
	cutilSafeCall (hipFree (f->trgNum_d_));
}







FMMWrapper_t *
preproc (AllNodes* All_N)
{
  FMMWrapper_t* f = (FMMWrapper_t *) malloc (sizeof (FMMWrapper_t));
  assert (f);

  f->AN = All_N;

	int i, j, idx;

	Point3 c;
	real_t r;
	// int num_leaf_nodes;
	// int num_non_leaf_nodes;

	int num_leaf_nodes_src;
	int num_leaf_nodes_trg;
	int num_non_leaf_nodes_src;
	int num_non_leaf_nodes_trg;

	int list_size;

	Pos *SP = All_N->SP;
	Trans_matrix *TM = All_N->TM;

	Pos *RP = All_N->RP;

  struct stopwatch_t* timer = NULL;
  struct stopwatch_t* timer_ = NULL;
  long double t_data_cpu, t_data_gpu, t_pcie, t_subtract;
  stopwatch_init ();
  timer = stopwatch_create ();
  timer_ = stopwatch_create ();


	/* ============================================================= */
	/* CPU SIDE 
	 */
	real_t* tmp_c;
	tmp_c = (real_t*) malloc (1024 * 1024);
	assert (tmp_c);

  fprintf (stderr, "Creating Host Data Structures ... ");
  stopwatch_start (timer);

	/* byte alignment required for coalesced loading */
	int byte_padding = get_byte_padding ();
	/* real_padding is padding in terms of # of data elements */
	int real_padding = byte_padding / sizeof (dtype);

	/* Create GPU friendly Source boxes */
	vector<NodeTree>& nodeVec = *All_N->N;
	/*
	ulist_create_boxes__double (&f->S_h_, nodeVec.size (), 
															All_N->Ns, real_padding);	
	 */
	ulist_create_boxes__double_source (All_N, f);

	/* Create GPU friendly Target boxes */
	/*
	ulist_create_boxes__double (&f->T_h_, nodeVec.size (), 
															All_N->Nt, real_padding);	
	 */
	ulist_create_boxes__double_target (All_N, f);

	/* Create GPU friendly ulist data structure */
	ulist_create_ulist (&f->U_h_, nodeVec.size (), All_N);	



	
	/* depth and children */
	f->depth_h_ = (int*) malloc (nodeVec.size () * sizeof (int));
	assert (f->depth_h_);
	for(i = 0; i < nodeVec.size (); i++) {
		f->depth_h_[i] = nodeVec[i].depth;
	}

	// num_leaf_nodes = 0;
	// num_non_leaf_nodes = 0;
	/*
	num_leaf_nodes = (int) pow (8.0, nodeVec[nodeVec.size () - 1].depth);	
	num_non_leaf_nodes = nodeVec.size () - num_leaf_nodes;
	assert ((num_leaf_nodes + num_non_leaf_nodes) == nodeVec.size ());
	 */
	/* num leaf and non-leaf nodes for src and trg */
	num_leaf_nodes_src = 0;
	num_non_leaf_nodes_src = 0;
	for(i = 0; i < nodeVec.size (); i++) {
		if(nodeVec[i].tag & LET_SRCNODE && nodeVec[i].child == -1) {
			num_leaf_nodes_src++; 
		} else if(nodeVec[i].tag & LET_SRCNODE && nodeVec[i].child != -1) {
			num_non_leaf_nodes_src++;
		} 
	}
	num_leaf_nodes_trg = 0;
	num_non_leaf_nodes_trg = 0;
	for(i = 0; i < nodeVec.size (); i++) {
		if(nodeVec[i].tag & LET_TRGNODE && nodeVec[i].child == -1) {
			num_leaf_nodes_trg++; 
		} else if(nodeVec[i].tag & LET_SRCNODE && nodeVec[i].child != -1) {
			num_non_leaf_nodes_trg++;
		} 
	}

	f->child_h_ = (int*) malloc (nodeVec.size () * sizeof (int));
	assert (f->child_h_);
	for(i = 0; i < nodeVec.size (); i++) {
		f->child_h_[i] = nodeVec[i].child;
	}


	/* Pre-compute center and radius */
	f->radius_h_ = (real_t*) malloc (nodeVec.size () * sizeof (real_t));
	f->center0_h_ = (real_t*) malloc (nodeVec.size () * sizeof (real_t));
	f->center1_h_ = (real_t*) malloc (nodeVec.size () * sizeof (real_t));
	f->center2_h_ = (real_t*) malloc (nodeVec.size () * sizeof (real_t));
	assert (f->radius_h_ && f->center0_h_ && f->center1_h_ && f->center2_h_);
	// idx = 0;
	for(i = 0; i < nodeVec.size (); i++) {

			c = center (i, nodeVec);
			r = radius (i, nodeVec);

			f->radius_h_[i] = r;
			f->center0_h_[i] = c(0);
			f->center1_h_[i] = c(1);
			f->center2_h_[i] = c(2);
	}


	/* tag */
	/* SRC or TG */
	f->tag_h_ = (int*) malloc (nodeVec.size () * sizeof (int));
	for(i = 0; i < nodeVec.size (); i++) {
		f->tag_h_[i] = nodeVec[i].tag;
	}

	/* SP[UC] */
	/* SP[UC] consists of 3 arrays x, y, and z each of which are
	 * (np+2)^3 - (np)^3 long
	 */
	/* allocate memory */
	f->SP_UC_size = pln_size (UC, SP);
	f->SP_UC_size_padded = (((pln_size (UC, SP) + real_padding - 1) / 
													real_padding) * real_padding);
	/* 3x for x, y, and z */
	f->SP_UC_h_ = (real_t*) malloc (3 * f->SP_UC_size_padded * sizeof (real_t));
	assert (f->SP_UC_h_);
	/* initialize data */
	memcpy (&f->SP_UC_h_[0], SP[UC].x, SP[UC].n * sizeof (real_t));
	memcpy (&f->SP_UC_h_[1 * f->SP_UC_size_padded], SP[UC].y, 
					SP[UC].n * sizeof (real_t));
	memcpy (&f->SP_UC_h_[2 * f->SP_UC_size_padded], SP[UC].z, 
					SP[UC].n * sizeof (real_t));

	/* UC2UE */
	stopwatch_start (timer_);
	compute_UC2UE_mat (TM, SP);	
	t_subtract = stopwatch_stop (timer_);

	f->UC2UE_r = pln_size (UE, SP);	
	f->UC2UE_r_padded = (((f->UC2UE_r + real_padding - 1) / real_padding) * 
											 real_padding);
	f->UC2UE_c = pln_size (UC, SP);	
	f->UC2UE_h_ = (real_t*) malloc (f->UC2UE_r_padded * f->UC2UE_c * 
																	sizeof (real_t));
	assert (f->UC2UE_h_);
	for(i = 0; i < f->UC2UE_c; i++) {
		memcpy (&f->UC2UE_h_[i * f->UC2UE_r_padded], &TM->UC2UE[i * f->UC2UE_r],
						f->UC2UE_r * sizeof (real_t));
	}


	/* UE2UC */
	stopwatch_start (timer_);
	TM->UE2UC = (real_t**) malloc (sizeof (real_t*) * 2 * 2 * 2);
	for(int a_ = 0; a_ < 2; a_++) {
		for(int b_ = 0; b_ < 2; b_++) {
			for(int c_ = 0; c_ < 2; c_++) {
				Index3 idx3(a_, b_, c_);
				compute_UE2UC_mat (idx3, TM, SP);
			}
		}	
	}
	t_subtract += stopwatch_stop (timer_);

	f->UE2UC_r = pln_size (UC, SP);
	f->UE2UC_r_padded = (((f->UE2UC_r + real_padding - 1) / real_padding) *
											 real_padding);
	f->UE2UC_c = pln_size (UE, SP);
	f->UE2UC_h_ = (real_t*) malloc ((2 * 2 * 2) * (f->UE2UC_r_padded * f->UE2UC_c)
																	*  sizeof (real_t));
	assert (f->UE2UC_h_);
	/* copy each matrix */
	for(i = 0; i < (2 * 2 * 2); i++) {
		/* 1 column at a time */
		for(j = 0; j < f->UE2UC_c; j++) {
			memcpy (&f->UE2UC_h_[i * f->UE2UC_r_padded * f->UE2UC_c + 
													 j * f->UE2UC_r_padded], 
							&TM->UE2UC[i][j * f->UE2UC_r], f->UE2UC_r * sizeof (dtype));
		}
	}

	/* SRC_UPW_EQU_DEN_h_ */
	f->SRC_UPW_EQU_DEN_h_ = (dtype*) malloc (nodeVec.size () * f->UC2UE_r_padded *
																					 sizeof (dtype));
	assert (f->SRC_UPW_EQU_DEN_h_);

	
	/* up_calc__gpu configuration variables */
	// f->num_non_leaf_nodes = num_non_leaf_nodes;
	f->tree_max_depth = nodeVec[nodeVec.size () - 1].depth;
	f->reduction_depth = f->tree_max_depth - 2;
	f->num_nodes_reduction = (int) pow (8.0, f->reduction_depth);
	f->reduction_offset = 0;
	for(i = 0; i < f->reduction_depth; i++) 
		f->reduction_offset += (int) pow (8.0, i);

	/* up_calc temporary arrays */
	/* src_upw_equ_den */
	/* There is no CPU equivalent of this as all this is needed is in the GPU */


	/* VLIST data structures */
	f->vlist_array_size = eff_data_size (UE);
	/*
	f->vlist_array_size_padded = (((f->vlist_array_size + real_padding - 1) / 
															 real_padding) * real_padding);
	 */
	/* trg */
	/* src */
	f->vlist_ptr_h_ = (int*) malloc ((nodeVec.size () + 1) * sizeof (int));
	assert (f->vlist_ptr_h_);

	list_size = 0;
	f->vlist_ptr_h_[0] = 0;
	for(i = 0; i < nodeVec.size (); i++) {
			list_size += nodeVec[i].Vnodes.size ();
			f->vlist_ptr_h_[i+1] = list_size;
	}
	f->vlist_h_ = (int*) malloc (list_size * sizeof (int));
	assert (f->vlist_h_);

	idx = 0;
	for(i = 0; i < nodeVec.size (); i++) {
		for(j = 0; j < nodeVec[i].Vnodes.size (); j++) {
			f->vlist_h_[idx] = nodeVec[i].Vnodes[j];
			idx++;
		}	
	}
	assert (idx == list_size);
	f->list_size = list_size;
	
	f->reg_den_size = RP->n;
	/*
	f->reg_den_size_padded = (((RP->n + real_padding - 1) / real_padding) * 
													 real_padding);
	 */
	/* reg den needs no host equivalent */
	
	/* trans */
	f->trans_arrays_num = 7 * 7 * 7;

	f->tlist_h_ = (int*) malloc (list_size * sizeof (int));
	assert (f->tlist_h_);
		
	int id;	
	int dim = 3;
	int t_index = 0;
	for(i = 0;i < nodeVec.size (); i++) {
		if(nodeVec[i].tag & LET_TRGNODE && nodeVec[i].Vnodes.size () > 0) {
			Point3 gNodeIdxCtr (center (i, nodeVec));
			real_t D = 2.0 * radius (i, nodeVec);
			for(j = 0;j < nodeVec[i].Vnodes.size (); j++) {
				idx = nodeVec[i].Vnodes[j];
				Point3 viCtr (center (idx, nodeVec));
				Index3 idx3;
				for(int d = 0; d < dim; d++) {
					idx3(d) = int (round ((viCtr[d] - gNodeIdxCtr[d]) / D));
				}
				id = (idx3(0) + 3) + (idx3(1) + 3) * 7 + (idx3(2) + 3) * 7 * 7;
				f->tlist_h_[t_index] = id;
				t_index++;
			}
		}
	}


	/* No need for these on the host */
	/* f->vlist_src_d_ */
	/* f->vlist_trg_d_ */
	/* f->vlist_trans_d_ */


	f->RP_n_ = RP->n;
	/* No need for these on the host */
	/* f->RP_X_d_ */
	/* f->RP_Y_d_ */
	/* f->RP_Z_d_ */


	/* IFFT */
	f->SP_DC_n_ = pln_size (DC, SP);
	f->SP_DC_n_padded_ = (((f->SP_DC_n_ + real_padding - 1) / real_padding) *
												real_padding);

	/* TRG_DWN_CHK_VAL_h_ */
	f->TRG_DWN_CHK_VAL_h_ = (dtype*) malloc (nodeVec.size () * 
																					 f->SP_DC_n_padded_ * sizeof (dtype));
	assert (f->TRG_DWN_CHK_VAL_h_);



	/* DOWN_CALC */
	/* path2Node */
	f->path2Node_h_ = (int3*) malloc (nodeVec.size () * sizeof (int3));
	assert (f->path2Node_h_);
	for(i = 0; i < nodeVec.size (); i++) {
		f->path2Node_h_[i].x = (nodeVec[i].path2Node)(0);
		f->path2Node_h_[i].y = (nodeVec[i].path2Node)(1);
		f->path2Node_h_[i].z = (nodeVec[i].path2Node)(2);
	}
	/* parent */	
	/* Not needed - use children structure instead */
	/* Actually, now I need it */
	f->parent_h_ = (int*) malloc (nodeVec.size () * sizeof (int));
	assert (f->parent_h_);
	for(i = 0; i < nodeVec.size (); i++) {
		f->parent_h_[i] = nodeVec[i].parent;
	}

	/* trg_dwn_equ_den */
	f->SP_DE_n_ = pln_size (DE, SP);	
	f->SP_DE_n_padded = (((f->SP_DE_n_ + real_padding - 1) / real_padding) *
											 real_padding);
	/* DC2DE_mat */
	stopwatch_start (timer_);
	compute_DC2DE_mat (TM, SP);
	t_subtract += stopwatch_stop (timer_);

	f->DC2DE_r = pln_size (DE, SP);
	f->DC2DE_r_padded = (((f->DC2DE_r + real_padding - 1) / real_padding) *
											 real_padding);
	f->DC2DE_c = pln_size (DC, SP);
	f->DC2DE_h_ = (real_t*) malloc (f->DC2DE_r_padded * f->DC2DE_c * 
																	sizeof (real_t));
	assert (f->DC2DE_h_);
	for(i = 0; i < f->DC2DE_c ; i++) {
		memcpy (&f->DC2DE_h_[i * f->DC2DE_r_padded],
						&TM->DC2DE[i * f->DC2DE_r], 
						f->DC2DE_r * sizeof (real_t));
	}
	/* DE2DC_mat[8] */
	stopwatch_start (timer_);
	TM->DE2DC = (real_t**) malloc (sizeof (real_t*) * 2 * 2 * 2);
	for(int a = 0; a < 2; a++) {
		for(int b = 0; b < 2; b++) {
			for(int c = 0; c < 2 ; c++) {
				Index3 idx(a, b, c);
				compute_DE2DC_mat (idx, TM, SP);
			}
		}
	}
	t_subtract += stopwatch_stop (timer_);

	f->DE2DC_r = pln_size (DC, SP);
	f->DE2DC_r_padded = (((f->DE2DC_r + real_padding - 1) / real_padding) *
											 real_padding);
	f->DE2DC_c = pln_size (DE, SP);
	f->DE2DC_h_ = (real_t*) malloc ((2 * 2 * 2) * f->DE2DC_r_padded * f->DE2DC_c *
																	sizeof (real_t));
	assert (f->DE2DC_h_);
	for(i = 0; i < 2 * 2 * 2; i++) {
		for(j = 0; j < f->DE2DC_c; j++) {
			real_t* temp_tm = TM->DE2DC[i];
			memcpy (&f->DE2DC_h_[i * f->DE2DC_r_padded * f->DE2DC_c + 
													 j * f->DE2DC_r_padded],
							&temp_tm[j * f->DE2DC_r],
							f->DE2DC_r * sizeof (real_t));
		}
	}
	/* down_calc configuration */
	f->expansion_depth = 2;
	f->num_nodes_expansion = (int) pow (8.0, f->expansion_depth);
	f->expansion_offset = 0;
	for(i = 0; i < f->expansion_depth; i++) {
		f->expansion_offset += (int) pow (8.0, i);
	}

	/* down_calc SP[DE] */
	f->SP_DE_h_ = (dtype*) malloc (3 * f->SP_DE_n_padded * sizeof (dtype));
	assert (f->SP_DE_h_);
	memcpy (&f->SP_DE_h_[0], SP[DE].x, SP[DE].n * sizeof (dtype));
	memcpy (&f->SP_DE_h_[f->SP_DE_n_padded], SP[DE].y, SP[DE].n * sizeof (dtype));
	memcpy (&f->SP_DE_h_[2 * f->SP_DE_n_padded], SP[DE].z, 
					SP[DE].n * sizeof (dtype));

  //t_data_cpu = stopwatch_stop (timer) - t_subtract;
  t_data_cpu = stopwatch_stop (timer); 
  fprintf (stderr, "==> Time: %Lg secs\n", t_data_cpu);


	/* WLIST_CALC */
	f->SP_UE_n_ = pln_size (UE, SP);	
	f->SP_UE_n_padded = (((f->SP_UE_n_ + real_padding - 1) / real_padding) *
											 real_padding);

	f->SP_UE_h_ = (dtype*) malloc (3 * f->SP_UE_n_padded * sizeof (dtype));
	assert (f->SP_UE_h_);
	memcpy (&f->SP_UE_h_[0], SP[UE].x, SP[UE].n * sizeof (dtype));
	memcpy (&f->SP_UE_h_[f->SP_UE_n_padded], SP[UE].y, SP[UE].n * sizeof (dtype));
	memcpy (&f->SP_UE_h_[2 * f->SP_UE_n_padded], SP[UE].z, 
					SP[UE].n * sizeof (dtype));

	wlist_create_wlist (&f->W_h_, nodeVec.size (), All_N);	


	f->srcNum_h_ = (int*) malloc (nodeVec.size () * sizeof (int));
	assert (f->srcNum_h_);
	for(i = 0; i < nodeVec.size (); i++) {
		f->srcNum_h_[i] = nodeVec[i].srcNum;
	}


	/* XLIST_CALC */
	f->SP_DC_h_ = (dtype*) malloc (3 * f->SP_DC_n_padded_ * sizeof (dtype));	
	assert (f->SP_DC_h_);

	memcpy (&f->SP_DC_h_[0], SP[DC].x, SP[DC].n * sizeof (dtype));
	memcpy (&f->SP_DC_h_[f->SP_DC_n_padded_], SP[DC].y, SP[DC].n * sizeof (dtype));
	memcpy (&f->SP_DC_h_[2 * f->SP_DC_n_padded_], SP[DC].z, 
					SP[DC].n * sizeof (dtype));

	xlist_create_xlist (&f->X_h_, nodeVec.size (), All_N);

	f->trgNum_h_ = (int*) malloc (nodeVec.size () * sizeof (int));
	assert (f->trgNum_h_);
	for(i = 0; i < nodeVec.size (); i++) {
		f->trgNum_h_[i] = nodeVec[i].trgNum;
	}




	#if 0
		long int bytes_up = 0;
		/* source boxes */
		bytes_up += 4 * f->S_h_.Bptr_[f->S_h_.n_boxes_] * sizeof (dtype);
		bytes_up += (f->S_h_.n_boxes_ + 1) * sizeof (int);
		bytes_up += (f->S_h_.n_boxes_) * sizeof (int);
	
		/* Radius */
		bytes_up += nodeVec.size () * sizeof (dtype);

		/* center */
		bytes_up += 3 * nodeVec.size () * sizeof (dtype);

		/* SP_UC */
		bytes_up += 3 * f->SP_UC_size_padded * sizeof (dtype);

		/* UC2UE */
		bytes_up += f->UC2UE_r_padded * f->UC2UE_c * sizeof (dtype);

		/* src_upw_equ_den */
		bytes_up += nodeVec.size () * f->UC2UE_r_padded * sizeof (dtype);

		/* child */
		bytes_up += nodeVec.size () * sizeof (int);

		/* UE2UC */
		bytes_up += 8 * f->UE2UC_r_padded * f->UE2UC_c * sizeof (dtype);

		/* tag */
		bytes_up += nodeVec.size () * sizeof (int);

		/* depth */
		bytes_up += nodeVec.size () * sizeof (int);

    double mega_bytes_up = (1.0 * bytes_up/ 1000000);
    printf("VLIST requires %g mega bytes of data\n", mega_bytes_up);
	#endif

	#if 0
    long int bytes_vlist = 0;
    /* DEPTH */
    bytes_vlist += nodeVec.size () * sizeof (int);
    /* SRC_UPW_EQU_DEN */
    bytes_vlist += nodeVec.size () * f->UC2UE_r_padded * sizeof (dtype);
    /* REG_DEN */
    bytes_vlist += nodeVec.size () * f->reg_den_size * sizeof (dtype);
    /* VLIST_SRC */
    bytes_vlist += nodeVec.size () * f->vlist_array_size * sizeof (dtype);
    /* TT */
    bytes_vlist += f->trans_arrays_num * f->RP_n_ * sizeof (dtype);
    /* VLIST_TRANS */
    bytes_vlist += f->trans_arrays_num * f->vlist_array_size * sizeof (dtype);
    /* VLIST_TRG */
    bytes_vlist += nodeVec.size () * f->vlist_array_size * sizeof (dtype);

    /* VLIST_TLIST */
    bytes_vlist += (nodeVec.size () + 1) * sizeof (int);
    bytes_vlist += list_size * sizeof (int);
    bytes_vlist += list_size * sizeof (int);

    /* REG_DEN_IFFT */
    bytes_vlist += nodeVec.size () * f->reg_den_size * sizeof (dtype);
    /* TRG_DWN_CHK_VAL */
    bytes_vlist += nodeVec.size () * f->SP_DC_n_padded_ * sizeof (dtype);

    double mega_bytes_vlist = (1.0 * bytes_vlist / 1000000);
    printf("VLIST requires %g mega bytes of data\n", mega_bytes_vlist);
	#endif


	/* ============================================================= */
	/* GPU SIDE 
	 */
	/* this is done to set up the GPU */
	real_t* tmp_g;
	cutilSafeCall (hipMalloc ((void**)&tmp_g, 1024 * 1024));
	
  fprintf (stderr, "Creating GPU Data Structures ... ");
  stopwatch_start (timer);



  #if __SOURCE_BOX__
	alloc__SOURCE_BOX__ (f);
  #endif

  #if __TARGET_BOX__
	alloc__TARGET_BOX__ (f);
  #endif

  #if __U_LIST__
	alloc__U_LIST__ (f);
  #endif

	#if __TAG__
	alloc__TAG__ (f);
	#endif

  #if __DEPTH__
	alloc__DEPTH__ (f);
  #endif

  #if __CHILDREN__
	alloc__CHILDREN__ (f);
  #endif
  
  #if __RADIUS__
	alloc__RADIUS__ (f);
  #endif

  #if __CENTER__
	alloc__CENTER__ (f);
  #endif

  #if __SP_UC__
	alloc__SP_UC__ (f);
  #endif

  #if __UC2UE__
	alloc__UC2UE__ (f);
  #endif

  #if __UE2UC__
	alloc__UE2UC__ (f);
  #endif

  #if __SRC_UPW_EQU_DEN__
	alloc__SRC_UPW_EQU_DEN__ (f);
  #endif

	/* Vlist */
  #if __VLIST_SRC__
	alloc__VLIST_SRC__ (f);
  #endif

  #if __REG_DEN__
	alloc__REG_DEN__ (f);
  #endif
  
  #if __TT__
	alloc__TT__ (f);
  #endif

  #if __VLIST_TRANS__
	alloc__VLIST_TRANS__ (f);
  #endif

  #if __VLIST_TRG__
	alloc__VLIST_TRG__ (f);
  #endif

  #if __VLIST_TLIST__
	alloc__VLIST_TLIST__ (f);
  #endif

	#if 0
  #if __REG_DEN_IFFT__
  /* ------------------------------------------------------------ */
	/* IFFT */	
	cutilSafeCall (hipMalloc ((void**)&f->reg_den_ifft_d_,
														 nodeVec.size () * f->reg_den_size *
														 sizeof (dtype)));
	assert (f->reg_den_ifft_d_);
  /* ------------------------------------------------------------ */
  #endif
	#endif
	
  #if __TRG_DWN_CHK_VAL__	
	alloc__TRG_DWN_CHK_VAL__ (f);
  #endif

	/* DOWN_CALC */
  #if __PATH2NODE__
	alloc__PATH2NODE__ (f);
  #endif

	#if __PARENT__
	alloc__PARENT__ (f);
	#endif

  #if __TRG_DWN_EQU_DEN__
	alloc__TRG_DWN_EQU_DEN__ (f);
  #endif

  #if __DC2DE__
	alloc__DC2DE__ (f);
  #endif

  #if __DE2DC__
	alloc__DE2DC__ (f);
  #endif

  #if __SP_DE__
	alloc__SP_DE__ (f);
  #endif

	#if __SP_UE__
	alloc__SP_UE__ (f);
	#endif


  #if __W_LIST__
	alloc__W_LIST__ (f);
  #endif

	#if __SRCNUM__
	alloc__SRCNUM__ (f);
	#endif

	#if __SP_DC__
	alloc__SP_DC__ (f);
	#endif


  #if __X_LIST__
	alloc__X_LIST__ (f);
  #endif

	#if __TRGNUM__
	alloc__TRGNUM__ (f);
	#endif

  t_data_gpu = stopwatch_stop (timer);
  fprintf (stderr, "==> Time: %Lg secs\n", t_data_gpu);


	/* ============================================================= */
	/* Copy data over to GPU 
	 */

  fprintf (stderr, "Copying Data over PCIE ... ");
  stopwatch_start (timer);

  #if __SOURCE_BOX__
	xfer__SOURCE_BOX__ (f);
  #endif

  #if __TARGET_BOX__
	xfer__TARGET_BOX__ (f);
  #endif

  #if __U_LIST__
	xfer__U_LIST__ (f);
  #endif

	#if __TAG__
	xfer__TAG__ (f);
	#endif

  #if __DEPTH__
	xfer__DEPTH__ (f);
  #endif

  #if __CHILDREN__
	xfer__CHILDREN__ (f);
  #endif

	/* center and radius */
  #if __RADIUS__
	xfer__RADIUS__ (f);
  #endif

  #if __CENTER__
	xfer__CENTER__ (f);
  #endif

	/* SP[UC] */
  #if __SP_UC__
	xfer__SP_UC__ (f);
  #endif

  #if __UC2UE__
	xfer__UC2UE__ (f);
  #endif
  
  #if __UE2UC__
	xfer__UE2UC__ (f);
  #endif

	/* No copying necessary for SRC_UPW_EQU_DEN_d_ */
	/* No copying necessary for vlist_src_d_, vlist_trg_d_, vlist_trans_d_*/
	/* No copying necessary for tt and reg_den */

	/* vlist, tlist and pointer */
  #if __VLIST_TLIST__
	xfer__VLIST_TLIST__ (f);
  #endif

	/* No copying necessary for reg_den_ifft_d_ */
	/* No copying necessary for TRG_DWN_CHK_VAL_d_ */

	/* DOWN_CALC */
	/* path2Node */
  #if __PATH2NODE__
	xfer__PATH2NODE__ (f);
  #endif

	/* No copying necessary for TRG_DWN_EQU_DEN_d_ */

	/* parent */
	/* Not needed - use children structure instead */
	/* Now I need it */
	#if __PARENT__
	xfer__PARENT__ (f);
	#endif

  #if __DC2DE__
	xfer__DC2DE__ (f);
  #endif

	/* DE2DC_mat[8] */
  #if __DE2DC__
	xfer__DE2DC__ (f);
  #endif

	/* SP[DE] */
  #if __SP_DE__
	xfer__SP_DE__ (f);
  #endif

	/* SP[UE] */
	#if __SP_UE__
	xfer__SP_UE__ (f);
	#endif

  #if __W_LIST__
	xfer__W_LIST__ (f);
  #endif

	#if __SRCNUM__
	xfer__SRCNUM__ (f);
	#endif


	#if __SP_DC__
	xfer__SP_DC__ (f);
	#endif

  #if __X_LIST__
	xfer__X_LIST__ (f);
  #endif

	#if __TRGNUM__
	xfer__TRGNUM__ (f);
	#endif


  t_pcie = stopwatch_stop (timer);
  fprintf (stderr, "==> Time: %Lg secs\n", t_pcie);



  return f;
}

