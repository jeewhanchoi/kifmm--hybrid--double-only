
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#include "evaluate.h"
#include "util.h"
#include "reals.h"
#include "partial.h"
#include "../timing/timing.h"
#include "node_gpu.h"
#include <cutil_inline.h>

int
run (FMMWrapper_t *F)
{
  AllNodes *All_N = F->AN;
  vector<NodeTree>& nodeVec = *All_N->N;

  struct stopwatch_t* timer = NULL;
  long double t_total;
  timer = stopwatch_create ();

    fprintf (stderr, "Performing Ulist on the GPU\n");
  stopwatch_start (timer);
    ulist_calc__gpu (F);
    up_calc__cpu (F);
    vlist_calc__cpu (F);
    hipDeviceSynchronize ();
		/* xfer up_calc result */
		/* first copy to SRC_UPW_EQU_DEN_h_ */
		for(int i = 0; i < nodeVec.size (); i++) {
			memcpy (&F->SRC_UPW_EQU_DEN_h_[i * F->UC2UE_r_padded], 
							&All_N->src_upw_equ_den[i * F->UC2UE_r], 
							F->UC2UE_r * sizeof (dtype));
		}
		/* first copy to SRC_UPW_EQU_DEN_h_ */
		for(int i = 0; i < nodeVec.size (); i++) {
			memcpy (&F->TRG_DWN_CHK_VAL_h_[i * F->SP_DC_n_padded_], 
							&All_N->trg_dwn_chk_val[i * F->SP_DC_n_], 
							F->SP_DC_n_ * sizeof (dtype));
		}
		/* DtoH copy of SRC_UPW_EQU_DEN */
		xfer__SRC_UPW_EQU_DEN__ (F);
		xfer__TRG_DWN_CHK_VAL__ (F);

		wlist_calc__gpu (F);
		xlist_calc__gpu (F);
    down_calc__gpu (F);
    copy_trg_val__gpu (F);
  t_total = stopwatch_stop (timer);
  
  fprintf(stderr, "=== Statistics ===\n");
  fprintf (stdout, "Code: %s\n", get_implementation_name ());
  fprintf (stdout, "Floating-point word size: %lu bytes\n", sizeof (real_t));
  fprintf (stderr, "== Timing for FMM3d ==\n");
  fprintf (stderr, "  ==> Total Execution Time: %Lg secs\n", t_total);

  stopwatch_destroy (timer);

  return 0;
}
/* ------------------------------------------------------------------------
 */
